#include "hip/hip_runtime.h"
#include <assert.h>
#include <errno.h>
#include <getopt.h>
#include <limits.h>
#include <math.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <utility>

#include "c63.h"
#include "common.h"
#include "init.h"
#include "init_cuda.h"
#include "me.h"
#include "sisci.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

extern "C" {
#include "tables.h"
}

/* getopt */
extern int optind;
extern char *optarg;


static void zero_out_prediction(struct c63_common* cm, const struct c63_cuda& c63_cuda)
{
	struct frame* frame = cm->curframe;
	hipMemsetAsync(frame->predicted_gpu->Y, 0, cm->ypw * cm->yph * sizeof(uint8_t),
			c63_cuda.streamY);
	hipMemsetAsync(frame->predicted_gpu->U, 0, cm->upw * cm->uph * sizeof(uint8_t),
			c63_cuda.streamU);
	hipMemsetAsync(frame->predicted_gpu->V, 0, cm->vpw * cm->vph * sizeof(uint8_t),
			c63_cuda.streamV);
}

static void c63_encode_image(struct c63_common *cm, const struct c63_common_gpu& cm_gpu,
		const struct c63_cuda& c63_cuda, struct segment_yuv* image_gpu)
{
	// Advance to next frame by swapping current and reference frame
	std::swap(cm->curframe, cm->refframe);

	cm->curframe->orig_gpu = image_gpu;

	/* Check if keyframe */
	if (cm->framenum == 0 || cm->frames_since_keyframe == cm->keyframe_interval)
	{
		cm->curframe->keyframe = 1;
		cm->frames_since_keyframe = 0;
	}
	else
	{
		cm->curframe->keyframe = 0;
	}

	if (!cm->curframe->keyframe)
	{
		/* Motion Estimation */
		gpu_c63_motion_estimate(cm, cm_gpu, c63_cuda);

		/* Motion Compensation */
		gpu_c63_motion_compensate(cm, c63_cuda);
	}
	else
	{
		// dct_quantize() expects zeroed out prediction buffers for key frames.
		// We zero them out here since we reuse the buffers from previous frames.
		zero_out_prediction(cm, c63_cuda);
	}

	yuv_t* predicted = cm->curframe->predicted_gpu;
	dct_t* residuals = cm->curframe->residuals_gpu;

	const dim3 threadsPerBlock(8, 8);

	const dim3 numBlocks_Y(cm->padw[Y_COMPONENT] / threadsPerBlock.x,
			cm->padh[Y_COMPONENT] / threadsPerBlock.y);
	const dim3 numBlocks_UV(cm->padw[U_COMPONENT] / threadsPerBlock.x,
			cm->padh[U_COMPONENT] / threadsPerBlock.y);

	/* DCT and Quantization */
	dct_quantize<<<numBlocks_Y, threadsPerBlock, 0, c63_cuda.streamY>>>(
			(const uint8_t*) cm->curframe->orig_gpu->Y, predicted->Y, cm->padw[Y_COMPONENT],
			residuals->Ydct, Y_COMPONENT);
	hipMemcpyAsync(cm->curframe->residuals->Ydct, residuals->Ydct,
			cm->padw[Y_COMPONENT] * cm->padh[Y_COMPONENT] * sizeof(int16_t), hipMemcpyDeviceToHost,
			c63_cuda.streamY);

	dct_quantize<<<numBlocks_UV, threadsPerBlock, 0, c63_cuda.streamU>>>(
			(const uint8_t*) cm->curframe->orig_gpu->U, predicted->U, cm->padw[U_COMPONENT],
			residuals->Udct, U_COMPONENT);
	hipMemcpyAsync(cm->curframe->residuals->Udct, residuals->Udct,
			cm->padw[U_COMPONENT] * cm->padh[U_COMPONENT] * sizeof(int16_t), hipMemcpyDeviceToHost,
			c63_cuda.streamU);

	dct_quantize<<<numBlocks_UV, threadsPerBlock, 0, c63_cuda.streamV>>>(
			(const uint8_t*) cm->curframe->orig_gpu->V, predicted->V, cm->padw[V_COMPONENT],
			residuals->Vdct, V_COMPONENT);
	hipMemcpyAsync(cm->curframe->residuals->Vdct, residuals->Vdct,
			cm->padw[V_COMPONENT] * cm->padh[V_COMPONENT] * sizeof(int16_t), hipMemcpyDeviceToHost,
			c63_cuda.streamV);

	/* Reconstruct frame for inter-prediction */
	dequantize_idct<<<numBlocks_Y, threadsPerBlock, 0, c63_cuda.streamY>>>(residuals->Ydct,
			predicted->Y, cm->ypw, cm->curframe->recons_gpu->Y, Y_COMPONENT);

	dequantize_idct<<<numBlocks_UV, threadsPerBlock, 0, c63_cuda.streamU>>>(residuals->Udct,
			predicted->U, cm->upw, cm->curframe->recons_gpu->U, U_COMPONENT);

	dequantize_idct<<<numBlocks_UV, threadsPerBlock, 0, c63_cuda.streamV>>>(residuals->Vdct,
			predicted->V, cm->vpw, cm->curframe->recons_gpu->V, V_COMPONENT);

	/* Function dump_image(), found in common.c, can be used here to check if the
	 prediction is correct */
}


struct c63_common* init_c63_enc(int width, int height, const struct c63_cuda& c63_cuda)
{
	/* calloc() sets allocated memory to zero */
	struct c63_common *cm = (struct c63_common*) calloc(1, sizeof(struct c63_common));

	cm->width = width;
	cm->height = height;

	cm->padw[Y_COMPONENT] = cm->ypw = (uint32_t) (ceil(width / 16.0f) * 16);
	cm->padh[Y_COMPONENT] = cm->yph = (uint32_t) (ceil(height / 16.0f) * 16);
	cm->padw[U_COMPONENT] = cm->upw = (uint32_t) (ceil(width * UX / (YX * 8.0f)) * 8);
	cm->padh[U_COMPONENT] = cm->uph = (uint32_t) (ceil(height * UY / (YY * 8.0f)) * 8);
	cm->padw[V_COMPONENT] = cm->vpw = (uint32_t) (ceil(width * VX / (YX * 8.0f)) * 8);
	cm->padh[V_COMPONENT] = cm->vph = (uint32_t) (ceil(height * VY / (YY * 8.0f)) * 8);

	cm->mb_colsY = cm->ypw / 8;
	cm->mb_colsU = cm->mb_colsY / 2;
	cm->mb_colsV = cm->mb_colsU;

	cm->mb_rowsY = cm->yph / 8;
	cm->mb_rowsU = cm->mb_rowsY / 2;
	cm->mb_rowsV = cm->mb_rowsU;

	/* Quality parameters -- Home exam deliveries should have original values,
	 i.e., quantization factor should be 25, search range should be 16, and the
	 keyframe interval should be 100. */
	cm->qp = 25;                  // Constant quantization factor. Range: [1..50]
	//cm->me_search_range = 16;   // This is now defined in c63.h
	cm->keyframe_interval = 100;  // Distance between keyframes

	/* Initialize quantization tables */
	for (int i = 0; i < 64; ++i)
	{
		cm->quanttbl[Y_COMPONENT][i] = yquanttbl_def[i] / (cm->qp / 10.0);
		cm->quanttbl[U_COMPONENT][i] = uvquanttbl_def[i] / (cm->qp / 10.0);
		cm->quanttbl[V_COMPONENT][i] = uvquanttbl_def[i] / (cm->qp / 10.0);
	}

	init_boundaries(cm, c63_cuda);

	cm->curframe = create_frame(cm, c63_cuda);
	cm->refframe = create_frame(cm, c63_cuda);

	return cm;
}

void free_c63_enc(struct c63_common* cm)
{
	cleanup_boundaries(cm);

	destroy_frame(cm->curframe);
	destroy_frame(cm->refframe);

	free(cm);
}

static void print_help()
{
	printf("Usage: ./c63enc [options]\n");
	printf("Command line options:\n");
	printf("  -a                             Local adapter number\n");
	printf("  -r                             Reader node ID\n");
	printf("  -w							 Writer node ID\n");
	printf("\n");

	exit(EXIT_FAILURE);
}

int main(int argc, char **argv)
{
	int c;

	if (argc == 1)
	{
		print_help();
	}

	unsigned int localAdapterNo = 0;
	unsigned int readerNodeId = 0;
	unsigned int writerNodeId = 0;

	while ((c = getopt(argc, argv, "a:r:w:")) != -1)
	{
		switch (c)
		{
			case 'a':
				localAdapterNo = atoi(optarg);
				break;
			case 'r':
				readerNodeId = atoi(optarg);
				break;
			case 'w':
				writerNodeId = atoi(optarg);
				break;
			default:
				print_help();
				break;
		}
	}

	if (optind > argc)
	{
		fprintf(stderr, "Error getting program options, try --help.\n");
		exit(EXIT_FAILURE);
	}

	/* Encode input frames */
	int numframes = 0;

	init_SISCI(localAdapterNo, readerNodeId, writerNodeId);

	uint32_t width, height;
	receive_width_and_height(&width, &height);
	send_width_and_height(width, height);

	struct c63_cuda c63_cuda = init_c63_cuda();
	struct c63_common *cm = init_c63_enc(width, height, c63_cuda);
	struct c63_common_gpu cm_gpu = init_c63_gpu(cm);

	set_sizes_offsets(cm);

	struct segment_yuv images_gpu[2];
	images_gpu[0] = init_image_segment(cm, 0);
	images_gpu[1] = init_image_segment(cm, 1);
	init_remote_encoded_data_segment(0);
	init_remote_encoded_data_segment(1);
	init_local_encoded_data_segments();

	//yuv_t* image_gpu = create_image_gpu(cm);
	int segNum = 0;

	int transferred = 0;
	while (1)
	{
		// The reader sends an interrupt when it has transferred the next frame
		int done = wait_for_reader(segNum);

		printf("Frame %d:", numframes);
		fflush(stdout);

		if (!done)
		{
			printf(" Received");
			fflush(stdout);
		}
		else
		{
			printf("\rNo more frames from reader\n");

			wait_for_writer(segNum^1);

			// Send interrupt to writer signaling that encoding has been finished
			signal_writer(ENCODING_FINISHED, segNum);
			break;
		}


		c63_encode_image(cm, cm_gpu, c63_cuda, &images_gpu[segNum]);

		// Wait until the GPU has finished encoding
		hipStreamSynchronize(c63_cuda.streamY);
		hipStreamSynchronize(c63_cuda.streamU);
		hipStreamSynchronize(c63_cuda.streamV);

		// Reader can transfer next frame
		signal_reader(segNum);

		printf(", encoded\n");
		fflush(stdout);

		wait_for_image_transfer(segNum);

		copy_to_segment(cm->curframe->mbs, cm->curframe->residuals, segNum);
		//cuda_copy_to_segment(cm, segNum);

		if (numframes >= NUM_IMAGE_SEGMENTS) {
			// The writer sends an interrupt when it is ready for the next frame
			wait_for_writer(segNum);
			//copy_to_segment(cm->curframe->mbs, cm->curframe->residuals, segNum);
			--transferred;
		}

		// Copy data frame to remote segment - interrupt to writer handled by callback

		transfer_encoded_data(cm->curframe->keyframe, segNum);
		++transferred;

		++cm->framenum;
		++cm->frames_since_keyframe;

		++numframes;

		segNum ^= 1;
	}

	//destroy_image_gpu(image_gpu);

	cleanup_c63_cuda(c63_cuda);
	free_c63_enc(cm);

	cleanup_segments();
	cleanup_SISCI();

	hipDeviceReset();

	return EXIT_SUCCESS;
}
