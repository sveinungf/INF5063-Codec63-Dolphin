#include "hip/hip_runtime.h"
#include <assert.h>
#include <errno.h>
#include <getopt.h>
#include <limits.h>
#include <math.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <utility>

#include "c63.h"
#include "common.h"
#include "me.h"
#include "sisci.h"

extern "C" {
#include "tables.h"
}


/* getopt */
extern int optind;
extern char *optarg;

static yuv_t image;

static void zero_out_prediction(struct c63_common* cm)
{
	struct frame* frame = cm->curframe;
	hipMemsetAsync(frame->predicted_gpu->Y, 0, cm->ypw * cm->yph * sizeof(uint8_t), cm->cuda_data.streamY);
	hipMemsetAsync(frame->predicted_gpu->U, 0, cm->upw * cm->uph * sizeof(uint8_t), cm->cuda_data.streamU);
	hipMemsetAsync(frame->predicted_gpu->V, 0, cm->vpw * cm->vph * sizeof(uint8_t), cm->cuda_data.streamV);
}

static void c63_encode_image(struct c63_common *cm, yuv_t* image_gpu)
{
	// Advance to next frame by swapping current and reference frame
	std::swap(cm->curframe, cm->refframe);

	cm->curframe->orig_gpu = image_gpu;

	/* Check if keyframe */
	if (cm->framenum == 0 || cm->frames_since_keyframe == cm->keyframe_interval)
	{
		cm->curframe->keyframe = 1;
		cm->frames_since_keyframe = 0;
	}
	else { cm->curframe->keyframe = 0; }

	if (!cm->curframe->keyframe)
	{
		/* Motion Estimation */
		c63_motion_estimate(cm);

		/* Motion Compensation */
		c63_motion_compensate(cm);
	}
	else
	{
		// dct_quantize() expects zeroed out prediction buffers for key frames.
		// We zero them out here since we reuse the buffers from previous frames.
		zero_out_prediction(cm);
	}

	yuv_t* predicted = cm->curframe->predicted_gpu;
	dct_t* residuals = cm->curframe->residuals_gpu;

	const dim3 threadsPerBlock(8, 8);

	const dim3 numBlocks_Y(cm->padw[Y_COMPONENT]/threadsPerBlock.x, cm->padh[Y_COMPONENT]/threadsPerBlock.y);
	const dim3 numBlocks_UV(cm->padw[U_COMPONENT]/threadsPerBlock.x, cm->padh[U_COMPONENT]/threadsPerBlock.y);

	/* DCT and Quantization */
	dct_quantize<<<numBlocks_Y, threadsPerBlock, 0, cm->cuda_data.streamY>>>(cm->curframe->orig_gpu->Y, predicted->Y,
			cm->padw[Y_COMPONENT], residuals->Ydct, Y_COMPONENT);
	hipMemcpyAsync(cm->curframe->residuals->Ydct, residuals->Ydct, cm->padw[Y_COMPONENT]*cm->padh[Y_COMPONENT]*sizeof(int16_t),
			hipMemcpyDeviceToHost, cm->cuda_data.streamY);

	dct_quantize<<<numBlocks_UV, threadsPerBlock, 0, cm->cuda_data.streamU>>>(cm->curframe->orig_gpu->U, predicted->U,
			cm->padw[U_COMPONENT], residuals->Udct, U_COMPONENT);
	hipMemcpyAsync(cm->curframe->residuals->Udct, residuals->Udct, cm->padw[U_COMPONENT]*cm->padh[U_COMPONENT]*sizeof(int16_t),
			hipMemcpyDeviceToHost, cm->cuda_data.streamU);

	dct_quantize<<<numBlocks_UV, threadsPerBlock, 0, cm->cuda_data.streamV>>>(cm->curframe->orig_gpu->V, predicted->V,
			cm->padw[V_COMPONENT], residuals->Vdct, V_COMPONENT);
	hipMemcpyAsync(cm->curframe->residuals->Vdct, residuals->Vdct, cm->padw[V_COMPONENT]*cm->padh[V_COMPONENT]*sizeof(int16_t),
			hipMemcpyDeviceToHost, cm->cuda_data.streamV);

	/* Reconstruct frame for inter-prediction */
	dequantize_idct<<<numBlocks_Y, threadsPerBlock, 0, cm->cuda_data.streamY>>>(residuals->Ydct, predicted->Y,
			cm->ypw, cm->curframe->recons_gpu->Y, Y_COMPONENT);

	dequantize_idct<<<numBlocks_UV, threadsPerBlock, 0, cm->cuda_data.streamU>>>(residuals->Udct, predicted->U,
			cm->upw, cm->curframe->recons_gpu->U, U_COMPONENT);

	dequantize_idct<<<numBlocks_UV, threadsPerBlock, 0, cm->cuda_data.streamV>>>(residuals->Vdct, predicted->V,
			cm->vpw, cm->curframe->recons_gpu->V, V_COMPONENT);

	/* Function dump_image(), found in common.c, can be used here to check if the
     prediction is correct */
}

static void init_boundaries(c63_common* cm)
{
	int hY = cm->padh[Y_COMPONENT];
	int hUV = cm->padh[U_COMPONENT];

	int wY = cm->padw[Y_COMPONENT];
	int wUV = cm->padw[U_COMPONENT];

	int* leftsY = new int[cm->mb_colsY];
	int* leftsUV = new int[cm->mb_colsUV];
	int* rightsY = new int[cm->mb_colsY];
	int* rightsUV = new int[cm->mb_colsUV];
	int* topsY = new int[cm->mb_rowsY];
	int* topsUV = new int[cm->mb_rowsUV];
	int* bottomsY = new int[cm->mb_rowsY];
	int* bottomsUV = new int[cm->mb_rowsUV];

	for (int mb_x = 0; mb_x < cm->mb_colsY; ++mb_x) {
		leftsY[mb_x] = mb_x * 8 - ME_RANGE_Y;
		rightsY[mb_x] = mb_x * 8 + ME_RANGE_Y;

		if (leftsY[mb_x] < 0) {
			leftsY[mb_x] = 0;
		}

		if (rightsY[mb_x] > (wY - 8)) {
			rightsY[mb_x] = wY - 8;
		}
	}

	for (int mb_x = 0; mb_x < cm->mb_colsUV; ++mb_x) {
		leftsUV[mb_x] = mb_x * 8 - ME_RANGE_UV;
		rightsUV[mb_x] = mb_x * 8 + ME_RANGE_UV;

		if (leftsUV[mb_x] < 0) {
			leftsUV[mb_x] = 0;
		}

		if (rightsUV[mb_x] > (wUV - 8)) {
			rightsUV[mb_x] = wUV - 8;
		}
	}

	for (int mb_y = 0; mb_y < cm->mb_rowsY; ++mb_y) {
		topsY[mb_y] = mb_y * 8 - ME_RANGE_Y;
		bottomsY[mb_y] = mb_y * 8 + ME_RANGE_Y;

		if (topsY[mb_y] < 0) {
			topsY[mb_y] = 0;
		}

		if (bottomsY[mb_y] > (hY - 8)) {
			bottomsY[mb_y] = hY - 8;
		}
	}

	for (int mb_y = 0; mb_y < cm->mb_rowsUV; ++mb_y) {
		topsUV[mb_y] = mb_y * 8 - ME_RANGE_UV;
		bottomsUV[mb_y] = mb_y * 8 + ME_RANGE_UV;

		if (topsUV[mb_y] < 0) {
			topsUV[mb_y] = 0;
		}

		if (bottomsUV[mb_y] > (hUV - 8)) {
			bottomsUV[mb_y] = hUV - 8;
		}
	}

	struct boundaries* boundY = &cm->me_boundariesY;
	hipMalloc((void**) &boundY->left, cm->mb_colsY * sizeof(int));
	hipMalloc((void**) &boundY->right, cm->mb_colsY * sizeof(int));
	hipMalloc((void**) &boundY->top, cm->mb_rowsY * sizeof(int));
	hipMalloc((void**) &boundY->bottom, cm->mb_rowsY * sizeof(int));

	struct boundaries* boundUV = &cm->me_boundariesUV;
	hipMalloc((void**) &boundUV->left, cm->mb_colsUV * sizeof(int));
	hipMalloc((void**) &boundUV->right, cm->mb_colsUV * sizeof(int));
	hipMalloc((void**) &boundUV->top, cm->mb_rowsUV * sizeof(int));
	hipMalloc((void**) &boundUV->bottom, cm->mb_rowsUV * sizeof(int));

	const hipStream_t& streamY = cm->cuda_data.streamY;
	hipMemcpyAsync((void*) boundY->left, leftsY, cm->mb_colsY * sizeof(int), hipMemcpyHostToDevice, streamY);
	hipMemcpyAsync((void*) boundY->right, rightsY, cm->mb_colsY * sizeof(int), hipMemcpyHostToDevice, streamY);
	hipMemcpyAsync((void*) boundY->top, topsY, cm->mb_rowsY * sizeof(int), hipMemcpyHostToDevice, streamY);
	hipMemcpyAsync((void*) boundY->bottom, bottomsY, cm->mb_rowsY * sizeof(int), hipMemcpyHostToDevice, streamY);

	hipMemcpy((void*) boundUV->left, leftsUV, cm->mb_colsUV * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy((void*) boundUV->right, rightsUV, cm->mb_colsUV * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy((void*) boundUV->top, topsUV, cm->mb_rowsUV * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy((void*) boundUV->bottom, bottomsUV, cm->mb_rowsUV * sizeof(int), hipMemcpyHostToDevice);

	delete[] leftsY;
	delete[] leftsUV;
	delete[] rightsY;
	delete[] rightsUV;
	delete[] topsY;
	delete[] topsUV;
	delete[] bottomsY;
	delete[] bottomsUV;
}

static void deinit_boundaries(c63_common* cm)
{
	hipFree((void*) cm->me_boundariesY.left);
	hipFree((void*) cm->me_boundariesY.right);
	hipFree((void*) cm->me_boundariesY.top);
	hipFree((void*) cm->me_boundariesY.bottom);

	hipFree((void*) cm->me_boundariesUV.left);
	hipFree((void*) cm->me_boundariesUV.right);
	hipFree((void*) cm->me_boundariesUV.top);
	hipFree((void*) cm->me_boundariesUV.bottom);
}

static void init_cuda_data(c63_common* cm)
{
	cuda_data* cuda_me = &(cm->cuda_data);

	hipStreamCreate(&cuda_me->streamY);
	hipStreamCreate(&cuda_me->streamU);
	hipStreamCreate(&cuda_me->streamV);

	hipMalloc((void**) &cuda_me->sad_index_resultsY, cm->mb_colsY*cm->mb_rowsY*sizeof(unsigned int));
	hipMalloc((void**) &cuda_me->sad_index_resultsU, cm->mb_colsUV*cm->mb_rowsUV*sizeof(unsigned int));
	hipMalloc((void**) &cuda_me->sad_index_resultsV, cm->mb_colsUV*cm->mb_rowsUV*sizeof(unsigned int));
}

static void deinit_cuda_data(c63_common* cm)
{
	hipStreamDestroy(cm->cuda_data.streamY);
	hipStreamDestroy(cm->cuda_data.streamU);
	hipStreamDestroy(cm->cuda_data.streamV);

	hipFree(cm->cuda_data.sad_index_resultsY);
	hipFree(cm->cuda_data.sad_index_resultsU);
	hipFree(cm->cuda_data.sad_index_resultsV);
}

static void copy_image_to_gpu(struct c63_common* cm, yuv_t* image, yuv_t* image_gpu)
{
	hipMemcpyAsync(image_gpu->Y, image->Y, cm->ypw * cm->yph * sizeof(uint8_t), hipMemcpyHostToDevice, cm->cuda_data.streamY);
	hipMemcpyAsync(image_gpu->U, image->U, cm->upw * cm->uph * sizeof(uint8_t), hipMemcpyHostToDevice, cm->cuda_data.streamU);
	hipMemcpyAsync(image_gpu->V, image->V, cm->vpw * cm->vph * sizeof(uint8_t), hipMemcpyHostToDevice, cm->cuda_data.streamV);
}

struct c63_common* init_c63_enc(int width, int height)
{
	/* calloc() sets allocated memory to zero */
	struct c63_common *cm = (struct c63_common*) calloc(1, sizeof(struct c63_common));

	cm->width = width;
	cm->height = height;

	cm->padw[Y_COMPONENT] = cm->ypw = (uint32_t) (ceil(width / 16.0f) * 16);
	cm->padh[Y_COMPONENT] = cm->yph = (uint32_t) (ceil(height / 16.0f) * 16);
	cm->padw[U_COMPONENT] = cm->upw = (uint32_t) (ceil(width * UX / (YX * 8.0f)) * 8);
	cm->padh[U_COMPONENT] = cm->uph = (uint32_t) (ceil(height * UY / (YY * 8.0f)) * 8);
	cm->padw[V_COMPONENT] = cm->vpw = (uint32_t) (ceil(width * VX / (YX * 8.0f)) * 8);
	cm->padh[V_COMPONENT] = cm->vph = (uint32_t) (ceil(height * VY / (YY * 8.0f)) * 8);

	cm->mb_colsY = cm->ypw / 8;
	cm->mb_rowsY = cm->yph / 8;
	cm->mb_colsUV = cm->mb_colsY / 2;
	cm->mb_rowsUV = cm->mb_rowsY / 2;

	/* Quality parameters -- Home exam deliveries should have original values,
	 i.e., quantization factor should be 25, search range should be 16, and the
	 keyframe interval should be 100. */
	cm->qp = 25;                  // Constant quantization factor. Range: [1..50]
	//cm->me_search_range = 16;   // This is now defined in c63.h
	cm->keyframe_interval = 100;  // Distance between keyframes

	/* Initialize quantization tables */
	for (int i = 0; i < 64; ++i)
	{
		cm->quanttbl[Y_COMPONENT][i] = yquanttbl_def[i] / (cm->qp / 10.0);
		cm->quanttbl[U_COMPONENT][i] = uvquanttbl_def[i] / (cm->qp / 10.0);
		cm->quanttbl[V_COMPONENT][i] = uvquanttbl_def[i] / (cm->qp / 10.0);
	}

	init_cuda_data(cm);

	cm->curframe = create_frame(cm);
	cm->refframe = create_frame(cm);

	init_boundaries(cm);

	return cm;
}

void free_c63_enc(struct c63_common* cm)
{
	deinit_boundaries(cm);

	destroy_frame(cm->curframe);
	destroy_frame(cm->refframe);

	deinit_cuda_data(cm);

	free(cm);
}

static void print_help()
{
	printf("Usage: ./c63enc [options]\n");
	printf("Command line options:\n");
	printf("  -a                             Local adapter number\n");
	printf("  -r                             Reader node ID\n");
	printf("  -w							 Writer node ID\n");
	printf("\n");

	exit(EXIT_FAILURE);
}

int main(int argc, char **argv)
{
	int c;

	if (argc == 1)
	{
		print_help();
	}

	unsigned int localAdapterNo = 0;
	unsigned int readerNodeId = 0;
	unsigned int writerNodeId = 0;

	while ((c = getopt(argc, argv, "a:r:w:")) != -1)
	{
		switch (c)
		{
			case 'a':
				localAdapterNo = atoi(optarg);
				break;
			case 'r':
				readerNodeId = atoi(optarg);
				break;
			case 'w':
				writerNodeId = atoi(optarg);
				break;
			default:
				print_help();
				break;
		}
	}

	if (optind > argc)
	{
		fprintf(stderr, "Error getting program options, try --help.\n");
		exit(EXIT_FAILURE);
	}

	/* Encode input frames */
	int numframes = 0;

	init_SISCI(localAdapterNo, readerNodeId, writerNodeId);

	uint32_t width, height;
	receive_width_and_height(&width, &height);
	send_width_and_height(width, height);

	struct c63_common *cm = init_c63_enc(width, height);

	image = init_image_segment(cm);
	init_encoded_data_segment(cm);

	yuv_t* image_gpu = create_image_gpu(cm);

	while (1)
	{
		printf("Frame %d:", numframes);
		fflush(stdout);

		// The reader sends an interrupt when it has transferred the next frame
		int done = wait_for_reader();

		if (!done)
		{
			printf(" Received");
			fflush(stdout);
		}
		else
		{
			printf("\rNo more frames from reader\n");

			// Send interrupt to writer signaling that encoding has been finished
			signal_writer(ENCODING_FINISHED);
			break;
		}

		copy_image_to_gpu(cm, &image, image_gpu);

		c63_encode_image(cm, image_gpu);

		// Wait until the GPU has finished encoding
		hipStreamSynchronize(cm->cuda_data.streamY);
		hipStreamSynchronize(cm->cuda_data.streamU);
		hipStreamSynchronize(cm->cuda_data.streamV);

		printf(", encoded");
		fflush(stdout);

		if (numframes != 0) {
			// The writer sends an interrupt when it is ready for the next frame
			wait_for_writer();
		}

		// Copy data frame to remote segment
		transfer_encoded_data(cm->curframe->keyframe, cm->curframe->mbs, cm->curframe->residuals);

		printf(", sent\n");

		// Send interrupt to writer signaling the data has been transfered
		signal_writer(DATA_TRANSFERRED);

		++cm->framenum;
		++cm->frames_since_keyframe;

		++numframes;

		// Reader can transfer next frame
		signal_reader();
	}

	destroy_image_gpu(image_gpu);

	free_c63_enc(cm);

	cleanup_segments();
	cleanup_SISCI();

	return EXIT_SUCCESS;
}
