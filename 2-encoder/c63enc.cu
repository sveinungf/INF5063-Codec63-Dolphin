#include "hip/hip_runtime.h"
#include <assert.h>
#include <errno.h>
#include <getopt.h>
#include <limits.h>
#include <math.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <utility>

#include "allocation.h"
#include "c63.h"
#include "init.h"
#include "sisci.h"
#include "cuda/common.h"
#include "cuda/init_cuda.h"
#include "cuda/me.h"

extern "C" {
#include "tables.h"
#include "simd/common.h"
#include "simd/me.h"
}


static const int Y = Y_COMPONENT;
static const int U = U_COMPONENT;
static const int V = V_COMPONENT;

/* getopt */
extern int optind;
extern char *optarg;

static void c63_encode_image(struct c63_common *cm, const struct c63_common_gpu& cm_gpu,
		const struct c63_cuda& c63_cuda, struct segment_yuv* image_gpu)
{
	// Advance to next frame by swapping current and reference frame
	std::swap(cm->curframe, cm->refframe);

	cm->curframe->orig_gpu = image_gpu;

	/* Check if keyframe */
	if (cm->framenum == 0 || cm->frames_since_keyframe == cm->keyframe_interval)
	{
		cm->curframe->keyframe = 1;
		cm->frames_since_keyframe = 0;
	}
	else
	{
		cm->curframe->keyframe = 0;
	}

	hipMemcpy(cm->curframe->orig->Y, (void*) cm->curframe->orig_gpu->Y, cm->ypw * cm->yph * sizeof(uint8_t), hipMemcpyDeviceToHost);
	hipMemcpy(cm->curframe->orig->U, (void*) cm->curframe->orig_gpu->U, cm->upw * cm->uph * sizeof(uint8_t), hipMemcpyDeviceToHost);
	hipMemcpy(cm->curframe->orig->V, (void*) cm->curframe->orig_gpu->V, cm->vpw * cm->vph * sizeof(uint8_t), hipMemcpyDeviceToHost);

	if (!cm->curframe->keyframe)
	{
		/* Motion Estimation */
		c63_motion_estimate_gpu(cm, cm_gpu, c63_cuda);
		c63_motion_estimate_host(cm);

		/* Motion Compensation */
		c63_motion_compensate_gpu(cm, c63_cuda);
		c63_motion_compensate_host(cm);
	}
	else
	{
		// dct_quantize() expects zeroed out prediction buffers for key frames.
		// We zero them out here since we reuse the buffers from previous frames.
		zero_out_prediction_gpu(cm, c63_cuda);
		zero_out_prediction_host(cm);
	}

	/* DCT and Quantization */
	dct_quantize_gpu(cm, c63_cuda);
	dct_quantize_host(cm);

	/* Reconstruct frame for inter-prediction */
	dequantize_idct_gpu(cm, c63_cuda);
	dequantize_idct_host(cm);

	/* Function dump_image(), found in common.c, can be used here to check if the
	 prediction is correct */
}

struct c63_common* init_c63_enc(int width, int height, const struct c63_cuda& c63_cuda)
{
	/* calloc() sets allocated memory to zero */
	struct c63_common *cm = (struct c63_common*) calloc(1, sizeof(struct c63_common));

	cm->width = width;
	cm->height = height;

	cm->padw[Y] = cm->ypw = (uint32_t) (ceil(width / 16.0f) * 16);
	cm->padh[Y] = cm->yph = (uint32_t) (ceil(height / 16.0f) * 16);
	cm->padw[U] = cm->upw = (uint32_t) (ceil(width * UX / (YX * 8.0f)) * 8);
	cm->padh[U] = cm->uph = (uint32_t) (ceil(height * UY / (YY * 8.0f)) * 8);
	cm->padw[V] = cm->vpw = (uint32_t) (ceil(width * VX / (YX * 8.0f)) * 8);
	cm->padh[V] = cm->vph = (uint32_t) (ceil(height * VY / (YY * 8.0f)) * 8);

	cm->mb_cols[Y] = cm->ypw / 8;
	cm->mb_cols[U] = cm->mb_cols[Y] / 2;
	cm->mb_cols[V] = cm->mb_cols[U];

	cm->mb_rows[Y] = cm->yph / 8;
	cm->mb_rows[U] = cm->mb_rows[Y] / 2;
	cm->mb_rows[V] = cm->mb_rows[U];

	/* Quality parameters -- Home exam deliveries should have original values,
	 i.e., quantization factor should be 25, search range should be 16, and the
	 keyframe interval should be 100. */
	cm->qp = 25;                  // Constant quantization factor. Range: [1..50]
	//cm->me_search_range = 16;   // This is now defined in c63.h
	cm->keyframe_interval = 100;  // Distance between keyframes

	/* Initialize quantization tables */
	for (int i = 0; i < 64; ++i)
	{
		cm->quanttbl[Y][i] = yquanttbl_def[i] / (cm->qp / 10.0);
		cm->quanttbl[U][i] = uvquanttbl_def[i] / (cm->qp / 10.0);
		cm->quanttbl[V][i] = uvquanttbl_def[i] / (cm->qp / 10.0);
	}

	init_me_boundaries(cm);

	cm->curframe = create_frame(cm, c63_cuda);
	cm->refframe = create_frame(cm, c63_cuda);

	return cm;
}

void free_c63_enc(struct c63_common* cm)
{
	cleanup_me_boundaries(cm);

	destroy_frame(cm->curframe);
	destroy_frame(cm->refframe);

	free(cm);
}

static void print_help()
{
	printf("Usage: ./c63enc [options]\n");
	printf("Command line options:\n");
	printf("  -a                             Local adapter number\n");
	printf("  -r                             Reader node ID\n");
	printf("  -w							 Writer node ID\n");
	printf("\n");

	exit(EXIT_FAILURE);
}

int main(int argc, char **argv)
{
	int c;

	if (argc == 1)
	{
		print_help();
	}

	unsigned int localAdapterNo = 0;
	unsigned int readerNodeId = 0;
	unsigned int writerNodeId = 0;

	while ((c = getopt(argc, argv, "a:r:w:")) != -1)
	{
		switch (c)
		{
			case 'a':
				localAdapterNo = atoi(optarg);
				break;
			case 'r':
				readerNodeId = atoi(optarg);
				break;
			case 'w':
				writerNodeId = atoi(optarg);
				break;
			default:
				print_help();
				break;
		}
	}

	if (optind > argc)
	{
		fprintf(stderr, "Error getting program options, try --help.\n");
		exit(EXIT_FAILURE);
	}

	/* Encode input frames */
	int numframes = 0;

	init_SISCI(localAdapterNo, readerNodeId, writerNodeId);

	uint32_t width, height;
	receive_width_and_height(&width, &height);
	send_width_and_height(width, height);

	struct c63_cuda c63_cuda = init_c63_cuda();
	struct c63_common *cm = init_c63_enc(width, height, c63_cuda);
	struct c63_common_gpu cm_gpu = init_c63_gpu(cm, c63_cuda);

	set_sizes_offsets(cm);

	struct segment_yuv images_gpu[2];
	images_gpu[0] = init_image_segment(cm, 0);
	init_remote_encoded_data_segment(0);
	init_remote_encoded_data_segment(1);
	init_local_encoded_data_segment();

	//yuv_t* image_gpu = create_image_gpu(cm);
	int segNum = 0;

	int transferred = 0;
	while (1)
	{
		// The reader sends an interrupt when it has transferred the next frame
		int done = wait_for_reader();

		printf("Frame %d:", numframes);
		fflush(stdout);

		if (!done)
		{
			printf(" Received");
			fflush(stdout);
		}
		else
		{
			printf("\rNo more frames from reader\n");

			wait_for_writer();

			// Send interrupt to writer signaling that encoding has been finished
			signal_writer(ENCODING_FINISHED);
			break;
		}

		c63_encode_image(cm, cm_gpu, c63_cuda, &images_gpu[0]);

		// Wait until the GPU has finished encoding
		hipStreamSynchronize(c63_cuda.stream[Y]);
		hipStreamSynchronize(c63_cuda.stream[U]);
		hipStreamSynchronize(c63_cuda.stream[V]);

		printf(", encoded\n");
		fflush(stdout);

		if (numframes != 0 && transferred == 2)
		{
			// The writer sends an interrupt when it is ready for the next frame
			wait_for_writer();
			--transferred;
		}

		// Copy data frame to remote segment - interrupt to writer handled by callback
		transfer_encoded_data(cm->curframe->keyframe, cm->curframe->mbs, cm->curframe->residuals,
				segNum);
		++transferred;

		// Reader can transfer next frame
		signal_reader();

		// Send interrupt to writer signaling the data has been transfered
		//signal_writer(DATA_TRANSFERRED);

		++cm->framenum;
		++cm->frames_since_keyframe;

		++numframes;

		segNum ^= 1;
	}

	//destroy_image_gpu(image_gpu);

	cleanup_c63_gpu(cm_gpu);
	free_c63_enc(cm);
	cleanup_c63_cuda(c63_cuda);

	cleanup_segments();
	cleanup_SISCI();

	return EXIT_SUCCESS;
}
