#include "init_cuda.h"


static const int Y = Y_COMPONENT;
static const int U = U_COMPONENT;
static const int V = V_COMPONENT;

static yuv_t* create_image_gpu(struct c63_common *cm)
{
	yuv_t* image = (yuv_t*) malloc(sizeof(yuv_t));
	hipMalloc((void**) &image->Y, cm->ypw * cm->yph * sizeof(uint8_t));
	hipMalloc((void**) &image->U, cm->upw * cm->uph * sizeof(uint8_t));
	hipMalloc((void**) &image->V, cm->vpw * cm->vph * sizeof(uint8_t));

	return image;
}

static void destroy_image_gpu(yuv_t* image)
{
	hipFree(image->Y);
	hipFree(image->U);
	hipFree(image->V);
	free(image);
}

void init_frame_gpu(struct c63_common* cm, struct frame* f)
{
	f->recons_gpu = create_image_gpu(cm);
	f->predicted_gpu = create_image_gpu(cm);

	f->residuals_gpu = (dct_t*) malloc(sizeof(dct_t));
	hipMalloc((void**) &f->residuals_gpu->Ydct, cm->ypw * cm->yph * sizeof(int16_t));
	hipMalloc((void**) &f->residuals_gpu->Udct, cm->upw * cm->uph * sizeof(int16_t));
	hipMalloc((void**) &f->residuals_gpu->Vdct, cm->vpw * cm->vph * sizeof(int16_t));

	hipMalloc((void**) &f->mbs_gpu[Y], cm->mb_rows[Y] * cm->mb_cols[Y] *
			sizeof(struct macroblock));
	hipMalloc((void**) &f->mbs_gpu[U], cm->mb_rows[U] * cm->mb_cols[U] *
			sizeof(struct macroblock));
	hipMalloc((void**) &f->mbs_gpu[V], cm->mb_rows[V] * cm->mb_cols[V] *
			sizeof(struct macroblock));
}

void deinit_frame_gpu(struct frame* f)
{
	destroy_image_gpu(f->recons_gpu);
	destroy_image_gpu(f->predicted_gpu);

	hipFree(f->residuals_gpu->Ydct);
	hipFree(f->residuals_gpu->Udct);
	hipFree(f->residuals_gpu->Vdct);
	free(f->residuals_gpu);

	hipFree(f->mbs_gpu[Y_COMPONENT]);
	hipFree(f->mbs_gpu[U_COMPONENT]);
	hipFree(f->mbs_gpu[V_COMPONENT]);
}

struct c63_cuda init_c63_cuda()
{
	struct c63_cuda result;

	for (int i = 0; i < COLOR_COMPONENTS; ++i)
	{
		hipStreamCreate(&result.stream[i]);
		hipStreamCreate(&result.memcpy_stream[i]);

		hipEventCreate(&result.me_done[i]);
		hipEventCreate(&result.dctquant_done[i]);
	}

	return result;
}

void cleanup_c63_cuda(struct c63_cuda& c63_cuda)
{
	for (int i = 0; i < COLOR_COMPONENTS; ++i)
	{
		hipStreamDestroy(c63_cuda.stream[i]);
		hipStreamDestroy(c63_cuda.memcpy_stream[i]);

		hipEventDestroy(c63_cuda.me_done[i]);
		hipEventDestroy(c63_cuda.dctquant_done[i]);
	}
}

static struct boundaries init_me_boundaries_gpu(const struct boundaries& indata, int cols, int rows,
		hipStream_t stream)
{
	struct boundaries result;

	hipMalloc((void**) &result.left, cols * sizeof(int));
	hipMalloc((void**) &result.right, cols * sizeof(int));
	hipMalloc((void**) &result.top, rows * sizeof(int));
	hipMalloc((void**) &result.bottom, rows * sizeof(int));

	hipMemcpyAsync((void*) result.left, indata.left, cols * sizeof(int), hipMemcpyHostToDevice,
			stream);
	hipMemcpyAsync((void*) result.right, indata.right, cols * sizeof(int), hipMemcpyHostToDevice,
			stream);
	hipMemcpyAsync((void*) result.top, indata.top, rows * sizeof(int), hipMemcpyHostToDevice,
			stream);
	hipMemcpyAsync((void*) result.bottom, indata.bottom, rows * sizeof(int),
			hipMemcpyHostToDevice, stream);

	return result;
}

static void cleanup_me_boundaries_gpu(struct boundaries& boundaries_gpu)
{
	hipFree((void*) boundaries_gpu.left);
	hipFree((void*) boundaries_gpu.right);
	hipFree((void*) boundaries_gpu.top);
	hipFree((void*) boundaries_gpu.bottom);
}

struct c63_common_gpu init_c63_gpu(const struct c63_common* cm, const struct c63_cuda& c63_cuda)
{
	struct c63_common_gpu result;

	for (int i = 0; i < COLOR_COMPONENTS; ++i)
	{
		int cols = cm->mb_cols[i];
		int rows = cm->mb_rows[i];
		const struct boundaries& boundaries = cm->me_boundaries[i];
		hipStream_t stream = c63_cuda.stream[i];

		result.me_boundaries[i] = init_me_boundaries_gpu(boundaries, cols, rows, stream);
		hipMalloc(&result.sad_index_results[i], cols * rows * sizeof(unsigned int));
	}

	return result;
}

void cleanup_c63_gpu(struct c63_common_gpu& cm_gpu)
{
	for (int i = 0; i < COLOR_COMPONENTS; ++i)
	{
		cleanup_me_boundaries_gpu(cm_gpu.me_boundaries[i]);
		hipFree(cm_gpu.sad_index_results[i]);
	}
}
