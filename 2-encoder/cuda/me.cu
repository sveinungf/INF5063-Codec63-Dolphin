#include "hip/hip_runtime.h"
#include <assert.h>
#include <errno.h>
#include <getopt.h>
#include <limits.h>
#include <math.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "c63_cuda.h"
#include "me.h"
#include "sisci_common.h"

namespace gpu = c63::gpu;


static const int Y = Y_COMPONENT;
static const int U = U_COMPONENT;
static const int V = V_COMPONENT;

__device__
static void min_warp_reduce(int i, volatile int* values)
{
	values[i] = min(values[i], values[i + 32]);
	values[i] = min(values[i], values[i + 16]);
	values[i] = min(values[i], values[i + 8]);
	values[i] = min(values[i], values[i + 4]);
	values[i] = min(values[i], values[i + 2]);
	values[i] = min(values[i], values[i + 1]);
}

template<int block_size>
__device__
static void min_reduce(int i, int* values)
{
	if (i < block_size/2)
	{
		// Intentionally no break between cases
		switch (block_size) {
			case 1024:
				values[i] = min(values[i], values[i + 512]);
				__syncthreads();
			case 512:
				values[i] = min(values[i], values[i + 256]);
				__syncthreads();
			case 256:
				values[i] = min(values[i], values[i + 128]);
				__syncthreads();
			case 128:
				values[i] = min(values[i], values[i + 64]);
				__syncthreads();
		}

		if (i < 32)
		{
			min_warp_reduce(i, values);
		}
	}
	else
	{
		switch (block_size) {
			case 1024:
				__syncthreads();
			case 512:
				__syncthreads();
			case 256:
				__syncthreads();
			case 128:
				__syncthreads();
		}
	}
}

template<int range>
__global__
static void me_block_8x8_gpu_Y(const uint8_t* __restrict__ orig, const uint8_t* __restrict__ ref,
		const int* __restrict__ lefts, const int* __restrict__ rights, const int* __restrict__ tops,
		const int* __restrict__ bottoms, int w, unsigned int* __restrict__ index_results)
{
	const int i = threadIdx.x;
	const int j = threadIdx.y;
	const int tid = j*blockDim.x + i;
	const int ref_mb_id = j*4*blockDim.x + i;
	const int ref_mb2_id = (j*4+1)*blockDim.x + i;
	const int ref_mb3_id = (j*4+2)*blockDim.x + i;
	const int ref_mb4_id = (j*4+3)*blockDim.x + i;

	const int mb_x = blockIdx.x;
	const int mb_y = blockIdx.y;
	const int orig_mb_id = mb_y*gridDim.x + mb_x;

	const int left = lefts[mb_x];
	const int top = tops[mb_y];
	const int right = rights[mb_x];
	const int bottom = bottoms[mb_y];

	const int mx = mb_x * 8;
	const int my = mb_y * 8;

	const uint8_t* orig_block = orig + my * w + mx;
	const uint8_t* ref_search_range = ref + top*w + left;

	__shared__ uint8_t shared_orig_block[64];

	if (i < 8 && j < 8)
	{
		shared_orig_block[j*8 + i] = orig_block[j*w + i];
	}

	__syncthreads();

	int block_sad = INT_MAX;
	int block2_sad = INT_MAX;
	int block3_sad = INT_MAX;
	int block4_sad = INT_MAX;

	const int range_width = right - left;
	const int range_height = (bottom - top)/4;

	const unsigned int mask = 0x3210 + 0x1111 * (i%4);

	// (i/4)*4 rounds i down to the nearest integer divisible by 4
	const uint8_t* ref_block_top_row_aligned = ref_search_range + (j*4)*w + (i/4)*4;

	if (j < range_height && i < range_width)
	{
		block_sad = 0;
		block2_sad = 0;
		block3_sad = 0;
		block4_sad = 0;

		#pragma unroll
		for (int y = 0; y < 8; ++y)
		{
			uint32_t* ref_block_row_aligned = (uint32_t*) (ref_block_top_row_aligned + y*w);
			uint32_t ref_row_left = __byte_perm(ref_block_row_aligned[0], ref_block_row_aligned[1], mask);
			uint32_t ref_row_right = __byte_perm(ref_block_row_aligned[1], ref_block_row_aligned[2], mask);

			uint32_t* ref_block2_row_aligned = (uint32_t*) (ref_block_top_row_aligned + (y+1)*w);
			uint32_t ref_row2_left = __byte_perm(ref_block2_row_aligned[0], ref_block2_row_aligned[1], mask);
			uint32_t ref_row2_right = __byte_perm(ref_block2_row_aligned[1], ref_block2_row_aligned[2], mask);

			uint32_t* ref_block3_row_aligned = (uint32_t*) (ref_block_top_row_aligned + (y+2)*w);
			uint32_t ref_row3_left = __byte_perm(ref_block3_row_aligned[0], ref_block3_row_aligned[1], mask);
			uint32_t ref_row3_right = __byte_perm(ref_block3_row_aligned[1], ref_block3_row_aligned[2], mask);

			uint32_t* ref_block4_row_aligned = (uint32_t*) (ref_block_top_row_aligned + (y+3)*w);
			uint32_t ref_row4_left = __byte_perm(ref_block4_row_aligned[0], ref_block4_row_aligned[1], mask);
			uint32_t ref_row4_right = __byte_perm(ref_block4_row_aligned[1], ref_block4_row_aligned[2], mask);

			uint8_t* orig_block_row = shared_orig_block + y*8;
			uint32_t orig_row_left = *((uint32_t*) orig_block_row);
			uint32_t orig_row_right = *((uint32_t*) orig_block_row + 1);

			block_sad += __vsadu4(ref_row_left, orig_row_left);
			block_sad += __vsadu4(ref_row_right, orig_row_right);

			block2_sad += __vsadu4(ref_row2_left, orig_row_left);
			block2_sad += __vsadu4(ref_row2_right, orig_row_right);

			block3_sad += __vsadu4(ref_row3_left, orig_row_left);
			block3_sad += __vsadu4(ref_row3_right, orig_row_right);

			block4_sad += __vsadu4(ref_row4_left, orig_row_left);
			block4_sad += __vsadu4(ref_row4_right, orig_row_right);
		}
	}

	__shared__ int block_sads[32*32];

	block_sads[ref_mb_id] = block_sad;
	block_sads[ref_mb2_id] = block2_sad;
	block_sads[ref_mb3_id] = block3_sad;
	block_sads[ref_mb4_id] = block4_sad;

	__syncthreads();

	block_sads[tid] = min(block_sads[tid], block_sads[tid + 512]);
	block_sads[tid + 256] = min(block_sads[tid + 256], block_sads[tid + 768]);
	__syncthreads();

	block_sads[tid] = min(block_sads[tid], block_sads[tid + 256]);
	__syncthreads();

	if (tid < 128) {
		block_sads[tid] = min(block_sads[tid], block_sads[tid + 128]);
	}
	__syncthreads();

	if (tid < 64) {
		block_sads[tid] = min(block_sads[tid], block_sads[tid + 64]);
	}
	__syncthreads();

	if (tid < 32) {
		min_warp_reduce(tid, block_sads);
	}
	__syncthreads();

	int min = block_sads[0];

	if (block_sad == min) {
		atomicMin(index_results + orig_mb_id, ref_mb_id);
	}

	if (block2_sad == min) {
		atomicMin(index_results + orig_mb_id, ref_mb2_id);
	}

	if (block3_sad == min) {
		atomicMin(index_results + orig_mb_id, ref_mb3_id);
	}

	if (block4_sad == min) {
		atomicMin(index_results + orig_mb_id, ref_mb4_id);
	}
}

template<int range>
__global__
static void me_block_8x8_gpu_UV(const uint8_t* __restrict__ orig, const uint8_t* __restrict__ ref,
		const int* __restrict__ lefts, const int* __restrict__ rights, const int* __restrict__ tops,
		const int* __restrict__ bottoms, int w, unsigned int* __restrict__ index_results)
{
	const int i = threadIdx.x;
	const int j = threadIdx.y;
	const int ref_mb_id = j*blockDim.x + i;

	const int mb_x = blockIdx.x;
	const int mb_y = blockIdx.y;
	const int orig_mb_id = mb_y*gridDim.x + mb_x;

	const int left = lefts[mb_x];
	const int top = tops[mb_y];
	const int right = rights[mb_x];
	const int bottom = bottoms[mb_y];

	const int mx = mb_x * 8;
	const int my = mb_y * 8;

	const uint8_t* orig_block = orig + my * w + mx;
	const uint8_t* ref_search_range = ref + top*w + left;

	__shared__ uint8_t shared_orig_block[64];

	if (i < 8 && j < 8)
	{
		shared_orig_block[j*8 + i] = orig_block[j*w + i];
	}

	__syncthreads();

	int block_sad = INT_MAX;

	const int range_width = right - left;
	const int range_height = bottom - top;

	const unsigned int mask = 0x3210 + 0x1111 * (i%4);

	// (i/4)*4 rounds i down to the nearest integer divisible by 4
	const uint8_t* ref_block_top_row_aligned = ref_search_range + j*w + (i/4)*4;

	if (j < range_height && i < range_width)
	{
		block_sad = 0;

		#pragma unroll
		for (unsigned int y = 8; y--; )
		{
			uint32_t* ref_block_row_aligned = (uint32_t*) (ref_block_top_row_aligned + y*w);
			uint32_t ref_row_left = __byte_perm(ref_block_row_aligned[0], ref_block_row_aligned[1], mask);
			uint32_t ref_row_right = __byte_perm(ref_block_row_aligned[1], ref_block_row_aligned[2], mask);

			uint8_t* orig_block_row = shared_orig_block + y*8;
			uint32_t orig_row_left = *((uint32_t*) orig_block_row);
			uint32_t orig_row_right = *((uint32_t*) orig_block_row + 1);

			block_sad += __vsadu4(ref_row_left, orig_row_left);
			block_sad += __vsadu4(ref_row_right, orig_row_right);
		}
	}

	const int max_range_width = range * 2;
	const int max_range_height = range * 2;
	const int max_mb_count = max_range_width * max_range_height;

	__shared__ int block_sads[max_mb_count];

	block_sads[ref_mb_id] = block_sad;

	__syncthreads();

	min_reduce<max_mb_count>(ref_mb_id, block_sads);

	__syncthreads();

	if (block_sad == block_sads[0]) {
		atomicMin(index_results + orig_mb_id, ref_mb_id);
	}
}

template<int range>
__global__
static void set_motion_vectors(struct macroblock* __restrict__ mbs, const int* __restrict__ lefts,
		const int* __restrict__ tops, const unsigned int* __restrict__ index_results)
{
	const int mb_x = blockIdx.x;
	const int mb_y = threadIdx.x;
	const int orig_mb_id = mb_y*gridDim.x + mb_x;

	const int left = lefts[mb_x];
	const int top = tops[mb_y];

	const int mx = mb_x * 8;
	const int my = mb_y * 8;

	int index_result = index_results[orig_mb_id];

	/* Here, there should be a threshold on SAD that checks if the motion vector
		 is cheaper than intraprediction. We always assume MV to be beneficial */
	struct macroblock* mb = &mbs[orig_mb_id];
	mb->use_mv = 1;
	mb->mv_x = left + (index_result % (range*2)) - mx;
	mb->mv_y = top + (index_result / (range*2)) - my;
}

template<int component>
void gpu::c63_motion_estimate(struct c63_common *cm, const struct c63_common_gpu& cm_gpu,
		const struct c63_cuda& c63_cuda)
{
	const int w = cm->padw[component];
	const int cols = cm->mb_cols[component];
	const int rows = cm->mb_rows[component];
	const int range = ME_RANGE(component);
	const struct boundaries& bound = cm_gpu.me_boundaries[component];
	const hipStream_t stream = c63_cuda.stream[component];

	unsigned int* sad_indexes = cm_gpu.sad_index_results[component];
	struct macroblock* mb = cm->curframe->mbs[component];
	struct macroblock* mb_gpu = cm->curframe->mbs_gpu[component];

	uint8_t* orig;
	uint8_t* ref;

	switch (component)
	{
		case Y_COMPONENT:
			orig = (uint8_t*) cm->curframe->orig_gpu->Y;
			ref = cm->refframe->recons_gpu->Y;
			break;
		case U_COMPONENT:
			orig = (uint8_t*) cm->curframe->orig_gpu->U;
			ref = cm->refframe->recons_gpu->U;
			break;
		case V_COMPONENT:
			orig = (uint8_t*) cm->curframe->orig_gpu->V;
			ref = cm->refframe->recons_gpu->V;
			break;
	}

	hipMemsetAsync(sad_indexes, 255, cols * rows * sizeof(unsigned int), stream);
	dim3 numBlocks(cols, rows);

	if (component == Y_COMPONENT)
	{
		// Luma
		dim3 threadsPerBlock(range * 2, range / 2);
		me_block_8x8_gpu_Y<range><<<numBlocks, threadsPerBlock, 0, stream>>>(orig, ref, bound.left, bound.right, bound.top, bound.bottom, w, sad_indexes);
	}
	else
	{
		// Chroma
		dim3 threadsPerBlock(range * 2, range * 2);
		me_block_8x8_gpu_UV<range><<<numBlocks, threadsPerBlock, 0, stream>>>(orig, ref, bound.left, bound.right, bound.top, bound.bottom, w, sad_indexes);
	}

	set_motion_vectors<range><<<cols, rows, 0, stream>>>(mb_gpu, bound.left, bound.top, sad_indexes);

	hipEvent_t me_done = c63_cuda.me_done[component];
	hipEventRecord(me_done, stream);

	hipStream_t memcpy_stream = c63_cuda.memcpy_stream[component];
	hipStreamWaitEvent(memcpy_stream, me_done, 0);
	hipMemcpyAsync(mb, mb_gpu, cols * rows * sizeof(struct macroblock), hipMemcpyDeviceToHost, memcpy_stream);
}

/* Motion compensation for 8x8 block */
__global__
static void mc_block_8x8_gpu(const struct macroblock* __restrict__ mbs, int w, uint8_t __restrict__ *predicted, const uint8_t __restrict__ *ref)
{
	const int mb_index = (blockIdx.x + blockIdx.y * gridDim.x);
	const int block_offset = mb_index * blockDim.x * blockDim.y;
	const int i = threadIdx.y;
	const int j = threadIdx.x;

	const struct macroblock* mb = &mbs[mb_index];

	// We always assume MV to be beneficial
	//if (!mb->use_mv) {
	//	return;
	//}

	const int mv_x = mb->mv_x;
	const int mv_y = mb->mv_y;

	/* Copy pixel from ref mandated by MV */
	predicted[block_offset + i * 8 + j] = ref[(i + blockIdx.y*8 + mv_y) * w + (j + blockIdx.x*8 + mv_x)];
}

template<int component>
void gpu::c63_motion_compensate(struct c63_common *cm, const struct c63_cuda& c63_cuda)
{
	const int w = cm->padw[component];
	const int h = cm->padh[component];
	const struct macroblock* mb = cm->curframe->mbs_gpu[component];
	const hipStream_t stream = c63_cuda.stream[component];

	uint8_t* pred;
	uint8_t* ref;

	switch (component)
	{
		case Y_COMPONENT:
			pred = cm->curframe->predicted_gpu->Y;
			ref = cm->refframe->recons_gpu->Y;
			break;
		case U_COMPONENT:
			pred = cm->curframe->predicted_gpu->U;
			ref = cm->refframe->recons_gpu->U;
			break;
		case V_COMPONENT:
			pred = cm->curframe->predicted_gpu->V;
			ref = cm->refframe->recons_gpu->V;
			break;
	}

	const dim3 threadsPerBlock(8, 8);
	const dim3 numBlocks(w / threadsPerBlock.x, h / threadsPerBlock.y);

	mc_block_8x8_gpu<<<numBlocks, threadsPerBlock, 0, stream>>>(mb, w, pred, ref);
}

template void gpu::c63_motion_estimate<Y>(struct c63_common *cm, const struct c63_common_gpu& cm_gpu,
		const struct c63_cuda& c63_cuda);
template void gpu::c63_motion_estimate<U>(struct c63_common *cm, const struct c63_common_gpu& cm_gpu,
		const struct c63_cuda& c63_cuda);
template void gpu::c63_motion_estimate<V>(struct c63_common *cm, const struct c63_common_gpu& cm_gpu,
		const struct c63_cuda& c63_cuda);

template void gpu::c63_motion_compensate<Y>(struct c63_common *cm, const struct c63_cuda& c63_cuda);
template void gpu::c63_motion_compensate<U>(struct c63_common *cm, const struct c63_cuda& c63_cuda);
template void gpu::c63_motion_compensate<V>(struct c63_common *cm, const struct c63_cuda& c63_cuda);
