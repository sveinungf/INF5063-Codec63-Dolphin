#include "init_cuda.h"

struct c63_cuda init_c63_cuda()
{
	struct c63_cuda result;

	for (int i = 0; i < COLOR_COMPONENTS; ++i)
	{
		hipStreamCreate(&result.stream[i]);
	}

	return result;
}

void cleanup_c63_cuda(struct c63_cuda& c63_cuda)
{
	for (int i = 0; i < COLOR_COMPONENTS; ++i)
	{
		hipStreamDestroy(c63_cuda.stream[i]);
	}
}

struct c63_common_gpu init_c63_gpu(struct c63_common* cm)
{
	struct c63_common_gpu result;

	for (int i = 0; i < COLOR_COMPONENTS; ++i)
	{
		size_t size = cm->mb_cols[i] * cm->mb_rows[i] * sizeof(unsigned int);
		hipMalloc(&result.sad_index_results[i], size);
	}

	return result;
}

void cleanup_c63_gpu(struct c63_common_gpu& cm_gpu)
{
	for (int i = 0; i < COLOR_COMPONENTS; ++i)
	{
		hipFree(cm_gpu.sad_index_results[i]);
	}
}

struct boundaries init_me_boundaries_gpu(const struct boundaries& indata, int cols, int rows,
		hipStream_t stream)
{
	struct boundaries result;

	hipMalloc((void**) &result.left, cols * sizeof(int));
	hipMalloc((void**) &result.right, cols * sizeof(int));
	hipMalloc((void**) &result.top, rows * sizeof(int));
	hipMalloc((void**) &result.bottom, rows * sizeof(int));

	hipMemcpyAsync((void*) result.left, indata.left, cols * sizeof(int), hipMemcpyHostToDevice,
			stream);
	hipMemcpyAsync((void*) result.right, indata.right, cols * sizeof(int), hipMemcpyHostToDevice,
			stream);
	hipMemcpyAsync((void*) result.top, indata.top, rows * sizeof(int), hipMemcpyHostToDevice,
			stream);
	hipMemcpyAsync((void*) result.bottom, indata.bottom, rows * sizeof(int),
			hipMemcpyHostToDevice, stream);

	return result;
}

void cleanup_me_boundaries_gpu(struct boundaries& boundaries_gpu)
{
	hipFree((void*) boundaries_gpu.left);
	hipFree((void*) boundaries_gpu.right);
	hipFree((void*) boundaries_gpu.top);
	hipFree((void*) boundaries_gpu.bottom);
}
