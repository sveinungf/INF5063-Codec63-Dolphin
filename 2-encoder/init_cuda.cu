#include "init_cuda.h"

struct c63_cuda init_c63_cuda()
{
	struct c63_cuda result;

	hipStreamCreate(&result.streamY);
	hipStreamCreate(&result.streamU);
	hipStreamCreate(&result.streamV);

	return result;
}

void cleanup_c63_cuda(struct c63_cuda& c63_cuda)
{
	hipStreamDestroy(c63_cuda.streamY);
	hipStreamDestroy(c63_cuda.streamU);
	hipStreamDestroy(c63_cuda.streamV);
}

struct c63_common_gpu init_c63_gpu(struct c63_common* cm)
{
	struct c63_common_gpu result;

	hipMalloc(&result.sad_index_resultsY, cm->mb_colsY * cm->mb_rowsY * sizeof(unsigned int));
	hipMalloc(&result.sad_index_resultsU, cm->mb_colsU * cm->mb_rowsU * sizeof(unsigned int));
	hipMalloc(&result.sad_index_resultsV, cm->mb_colsV * cm->mb_rowsV * sizeof(unsigned int));

	return result;
}

void cleanup_c63_gpu(struct c63_common_gpu& cm_gpu)
{
	hipFree(cm_gpu.sad_index_resultsY);
	hipFree(cm_gpu.sad_index_resultsU);
	hipFree(cm_gpu.sad_index_resultsV);
}

struct boundaries init_me_boundaries_gpu(const struct boundaries& indata, int cols, int rows,
		hipStream_t stream)
{
	struct boundaries result;

	hipMalloc((void**) &result.left, cols * sizeof(int));
	hipMalloc((void**) &result.right, cols * sizeof(int));
	hipMalloc((void**) &result.top, rows * sizeof(int));
	hipMalloc((void**) &result.bottom, rows * sizeof(int));

	hipMemcpyAsync((void*) result.left, indata.left, cols * sizeof(int), hipMemcpyHostToDevice,
			stream);
	hipMemcpyAsync((void*) result.right, indata.right, cols * sizeof(int), hipMemcpyHostToDevice,
			stream);
	hipMemcpyAsync((void*) result.top, indata.top, rows * sizeof(int), hipMemcpyHostToDevice,
			stream);
	hipMemcpyAsync((void*) result.bottom, indata.bottom, rows * sizeof(int),
			hipMemcpyHostToDevice, stream);

	return result;
}

void cleanup_me_boundaries_gpu(struct boundaries& boundaries_gpu)
{
	hipFree((void*) boundaries_gpu.left);
	hipFree((void*) boundaries_gpu.right);
	hipFree((void*) boundaries_gpu.top);
	hipFree((void*) boundaries_gpu.bottom);
}
