#include "init_cuda.h"

struct c63_cuda init_c63_cuda()
{
	struct c63_cuda result;

	for (int i = 0; i < COLOR_COMPONENTS; ++i)
	{
		hipStreamCreate(&result.stream[i]);
	}

	return result;
}

void cleanup_c63_cuda(struct c63_cuda& c63_cuda)
{
	for (int i = 0; i < COLOR_COMPONENTS; ++i)
	{
		hipStreamDestroy(c63_cuda.stream[i]);
	}
}

static struct boundaries init_me_boundaries_gpu(const struct boundaries& indata, int cols, int rows,
		hipStream_t stream)
{
	struct boundaries result;

	hipMalloc((void**) &result.left, cols * sizeof(int));
	hipMalloc((void**) &result.right, cols * sizeof(int));
	hipMalloc((void**) &result.top, rows * sizeof(int));
	hipMalloc((void**) &result.bottom, rows * sizeof(int));

	hipMemcpyAsync((void*) result.left, indata.left, cols * sizeof(int), hipMemcpyHostToDevice,
			stream);
	hipMemcpyAsync((void*) result.right, indata.right, cols * sizeof(int), hipMemcpyHostToDevice,
			stream);
	hipMemcpyAsync((void*) result.top, indata.top, rows * sizeof(int), hipMemcpyHostToDevice,
			stream);
	hipMemcpyAsync((void*) result.bottom, indata.bottom, rows * sizeof(int),
			hipMemcpyHostToDevice, stream);

	return result;
}

static void cleanup_me_boundaries_gpu(struct boundaries& boundaries_gpu)
{
	hipFree((void*) boundaries_gpu.left);
	hipFree((void*) boundaries_gpu.right);
	hipFree((void*) boundaries_gpu.top);
	hipFree((void*) boundaries_gpu.bottom);
}

struct c63_common_gpu init_c63_gpu(const struct c63_common* cm, const struct c63_cuda& c63_cuda)
{
	struct c63_common_gpu result;

	for (int i = 0; i < COLOR_COMPONENTS; ++i)
	{
		int cols = cm->mb_cols[i];
		int rows = cm->mb_rows[i];
		const struct boundaries& boundaries = cm->me_boundaries[i];
		hipStream_t stream = c63_cuda.stream[i];

		result.me_boundaries[i] = init_me_boundaries_gpu(boundaries, cols, rows, stream);
		hipMalloc(&result.sad_index_results[i], cols * rows * sizeof(unsigned int));
	}

	return result;
}

void cleanup_c63_gpu(struct c63_common_gpu& cm_gpu)
{
	for (int i = 0; i < COLOR_COMPONENTS; ++i)
	{
		cleanup_me_boundaries_gpu(cm_gpu.me_boundaries[i]);
		hipFree(cm_gpu.sad_index_results[i]);
	}
}
