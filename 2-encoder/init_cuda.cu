#include "init_cuda.h"

struct boundaries init_me_boundaries_gpu(struct boundaries* indata, int cols, int rows, hipStream_t stream)
{
	struct boundaries result;

	hipMalloc((void**) &result.left, cols * sizeof(int));
	hipMalloc((void**) &result.right, cols * sizeof(int));
	hipMalloc((void**) &result.top, rows * sizeof(int));
	hipMalloc((void**) &result.bottom, rows * sizeof(int));

	hipMemcpyAsync((void*) result.left, indata->left, cols * sizeof(int), hipMemcpyHostToDevice, stream);
	hipMemcpyAsync((void*) result.right, indata->right, cols * sizeof(int), hipMemcpyHostToDevice, stream);
	hipMemcpyAsync((void*) result.top, indata->top, rows * sizeof(int), hipMemcpyHostToDevice, stream);
	hipMemcpyAsync((void*) result.bottom, indata->bottom, rows * sizeof(int), hipMemcpyHostToDevice, stream);

	return result;
}

void cleanup_me_boundaries_gpu(struct boundaries* boundaries_gpu)
{
	hipFree((void*) boundaries_gpu->left);
	hipFree((void*) boundaries_gpu->right);
	hipFree((void*) boundaries_gpu->top);
	hipFree((void*) boundaries_gpu->bottom);
}
