#include "init_cuda.h"

struct c63_cuda init_c63_cuda()
{
	struct c63_cuda result;

	for (int i = 0; i < COLOR_COMPONENTS; ++i)
	{
		hipStreamCreate(&result.stream[i]);
	}

	return result;
}

void cleanup_c63_cuda(struct c63_cuda& c63_cuda)
{
	for (int i = 0; i < COLOR_COMPONENTS; ++i)
	{
		hipStreamDestroy(c63_cuda.stream[i]);
	}
}

struct c63_common_gpu init_c63_gpu(struct c63_common* cm)
{
	static const int Y = Y_COMPONENT;
	static const int U = U_COMPONENT;
	static const int V = V_COMPONENT;

	struct c63_common_gpu result;

	hipMalloc(&result.sad_index_resultsY, cm->mb_cols[Y] * cm->mb_rows[Y] * sizeof(unsigned int));
	hipMalloc(&result.sad_index_resultsU, cm->mb_cols[U] * cm->mb_rows[U] * sizeof(unsigned int));
	hipMalloc(&result.sad_index_resultsV, cm->mb_cols[V] * cm->mb_rows[V] * sizeof(unsigned int));

	return result;
}

void cleanup_c63_gpu(struct c63_common_gpu& cm_gpu)
{
	hipFree(cm_gpu.sad_index_resultsY);
	hipFree(cm_gpu.sad_index_resultsU);
	hipFree(cm_gpu.sad_index_resultsV);
}

struct boundaries init_me_boundaries_gpu(const struct boundaries& indata, int cols, int rows,
		hipStream_t stream)
{
	struct boundaries result;

	hipMalloc((void**) &result.left, cols * sizeof(int));
	hipMalloc((void**) &result.right, cols * sizeof(int));
	hipMalloc((void**) &result.top, rows * sizeof(int));
	hipMalloc((void**) &result.bottom, rows * sizeof(int));

	hipMemcpyAsync((void*) result.left, indata.left, cols * sizeof(int), hipMemcpyHostToDevice,
			stream);
	hipMemcpyAsync((void*) result.right, indata.right, cols * sizeof(int), hipMemcpyHostToDevice,
			stream);
	hipMemcpyAsync((void*) result.top, indata.top, rows * sizeof(int), hipMemcpyHostToDevice,
			stream);
	hipMemcpyAsync((void*) result.bottom, indata.bottom, rows * sizeof(int),
			hipMemcpyHostToDevice, stream);

	return result;
}

void cleanup_me_boundaries_gpu(struct boundaries& boundaries_gpu)
{
	hipFree((void*) boundaries_gpu.left);
	hipFree((void*) boundaries_gpu.right);
	hipFree((void*) boundaries_gpu.top);
	hipFree((void*) boundaries_gpu.bottom);
}
