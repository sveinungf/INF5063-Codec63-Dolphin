#include <sisci_api.h>
#include <sisci_error.h>

#include "../common/sisci_errchk.h"
#include "sisci.h"

#define MIN_SEG_SZ 237569
#define LOK 2048

// Local
static unsigned int localAdapterNo;
static unsigned int localNodeId;
sci_desc_t reader_sds[NUM_IMAGE_SEGMENTS];
sci_desc_t writer_sds[NUM_IMAGE_SEGMENTS];
uint8_t* cudaBuffers[NUM_IMAGE_SEGMENTS];

// Reader
unsigned int readerNodeId;
sci_local_data_interrupt_t interruptsFromReader[NUM_IMAGE_SEGMENTS];
sci_remote_interrupt_t interruptsToReader[NUM_IMAGE_SEGMENTS];
sci_local_segment_t imageSegments[NUM_IMAGE_SEGMENTS] __attribute__((aligned(sizeof(sci_local_segment_t))));
sci_map_t imageMaps[NUM_IMAGE_SEGMENTS] __attribute__((aligned(sizeof(sci_map_t))));

// Writer
static unsigned int segmentSizeWriter;
static unsigned int writerNodeId;
static sci_local_interrupt_t interruptsFromWriter[NUM_IMAGE_SEGMENTS];
static sci_remote_data_interrupt_t interruptsToWriter[NUM_IMAGE_SEGMENTS];
static sci_remote_segment_t encodedDataSegmentsWriter[NUM_IMAGE_SEGMENTS];
static sci_local_segment_t encodedDataSegmentsLocal[NUM_IMAGE_SEGMENTS];
static sci_map_t encodedDataMapsLocal[NUM_IMAGE_SEGMENTS];
static sci_dma_queue_t dmaQueues[NUM_IMAGE_SEGMENTS];

unsigned int keyframeSize;
unsigned int mbSizeY;
unsigned int mbSizeU;
unsigned int mbSizeV;
unsigned int residualsSizeY;
unsigned int residualsSizeU;
unsigned int residualsSizeV;

unsigned int keyframe_offset;
unsigned int mbOffsetY;
unsigned int residualsY_offset;
unsigned int mbOffsetU;
unsigned int residualsU_offset;
unsigned int mbOffsetV;
unsigned int residualsV_offset;

volatile int *keyframe[NUM_IMAGE_SEGMENTS];
struct macroblock *mb_Y[NUM_IMAGE_SEGMENTS];
struct macroblock *mb_U[NUM_IMAGE_SEGMENTS];
struct macroblock *mb_V[NUM_IMAGE_SEGMENTS];


volatile struct macroblock *remote_mb_Y[NUM_IMAGE_SEGMENTS];
volatile struct macroblock *remote_mb_U[NUM_IMAGE_SEGMENTS];
volatile struct macroblock *remote_mb_V[NUM_IMAGE_SEGMENTS];
volatile uint8_t *remote_residuals_Y[NUM_IMAGE_SEGMENTS];
volatile uint8_t *remote_residuals_U[NUM_IMAGE_SEGMENTS];
volatile uint8_t *remote_residuals_V[NUM_IMAGE_SEGMENTS];
static sci_map_t encodedDataMapsRemote[NUM_IMAGE_SEGMENTS];


void init_SISCI(unsigned int localAdapter, unsigned int readerNode, unsigned int writerNode)
{
	localAdapterNo = localAdapter;
	readerNodeId = readerNode;
	writerNodeId = writerNode;

	sci_error_t error;

	SCIInitialize(SCI_NO_FLAGS, &error);
	sisci_assert(error);

	int i;
	for (i = 0; i < NUM_IMAGE_SEGMENTS; ++i) {
		SCIOpen(&reader_sds[i], SCI_NO_FLAGS, &error);
		sisci_assert(error);

		SCIOpen(&writer_sds[i], SCI_NO_FLAGS, &error);
		sisci_assert(error);
	}

	SCIGetLocalNodeId(localAdapterNo, &localNodeId, SCI_NO_FLAGS, &error);
	sisci_assert(error);

	unsigned int maxEntries = 1;
	for (i = 0; i < NUM_IMAGE_SEGMENTS; ++i) {
		SCICreateDMAQueue(writer_sds[i], &dmaQueues[i], localAdapterNo, maxEntries, SCI_NO_FLAGS, &error);
		sisci_assert(error);
	}

	// Interrupts from the reader
	unsigned int interruptFromReaderNo;
	for (i = 0; i < NUM_IMAGE_SEGMENTS; ++i) {
		interruptFromReaderNo = MORE_DATA_TRANSFERRED + i;
		SCICreateDataInterrupt(reader_sds[i], &interruptsFromReader[i], localAdapterNo, &interruptFromReaderNo, NULL,
				NULL, SCI_FLAG_FIXED_INTNO, &error);
		sisci_assert(error);
	}

	// Interrupts from the writer
	unsigned int interruptFromWriterNo;
	for (i = 0; i < NUM_IMAGE_SEGMENTS; ++i) {
		interruptFromWriterNo = DATA_WRITTEN + i;
		SCICreateInterrupt(writer_sds[i], &interruptsFromWriter[i], localAdapterNo, &interruptFromWriterNo, NULL,
				NULL, SCI_FLAG_FIXED_INTNO, &error);
		sisci_assert(error);
	}

	// Interrupts to the reader
	printf("Connecting to interrupts on reader... ");
	fflush(stdout);
	for (i = 0; i < NUM_IMAGE_SEGMENTS; ++i) {
		do
		{
			SCIConnectInterrupt(reader_sds[i], &interruptsToReader[i], readerNodeId, localAdapterNo,
					READY_FOR_ORIG_TRANSFER + i, SCI_INFINITE_TIMEOUT, SCI_NO_FLAGS, &error);
		}
		while (error != SCI_ERR_OK);
	}
	printf("Done!\n");

	// Interrupts to the writer
	printf("Connecting to interrupt on writer... ");
	fflush(stdout);
	for (i = 0; i < NUM_IMAGE_SEGMENTS; ++i) {
		do
		{
			SCIConnectDataInterrupt(writer_sds[i], &interruptsToWriter[i], writerNodeId, localAdapterNo,
					ENCODED_FRAME_TRANSFERRED + i, SCI_INFINITE_TIMEOUT, SCI_NO_FLAGS, &error);
		}
		while (error != SCI_ERR_OK);
	}
	printf("Done!\n");
}

void cleanup_SISCI()
{
	sci_error_t error;

	int i;
	for (i = 0; i < NUM_IMAGE_SEGMENTS; ++i) {
		SCIDisconnectDataInterrupt(interruptsToWriter[i], SCI_NO_FLAGS, &error);
		sisci_check(error);
		SCIDisconnectInterrupt(interruptsToReader[i], SCI_NO_FLAGS, &error);
		sisci_check(error);

		do {
				SCIRemoveInterrupt(interruptsFromWriter[i], SCI_NO_FLAGS, &error);
		} while (error != SCI_ERR_OK);

		do {
			SCIRemoveDataInterrupt(interruptsFromReader[i], SCI_NO_FLAGS, &error);
		} while (error != SCI_ERR_OK);

		SCIRemoveDMAQueue(dmaQueues[i], SCI_NO_FLAGS, &error);

		SCIClose(reader_sds[i], SCI_NO_FLAGS, &error);
		sisci_check(error);

		SCIClose(writer_sds[i], SCI_NO_FLAGS, &error);
		sisci_check(error);
	}
	SCITerminate();
}


void set_sizes_offsets(struct c63_common *cm) {
    keyframeSize = sizeof(int);
    mbSizeY = cm->mb_rowsY * cm->mb_colsY * sizeof(struct macroblock);
    mbSizeU = cm->mb_rowsUV * cm->mb_colsUV * sizeof(struct macroblock);
    mbSizeV = cm->mb_rowsUV * cm->mb_colsUV * sizeof(struct macroblock);
    residualsSizeY = cm->ypw * cm->yph * sizeof(int16_t);
    residualsSizeU = cm->upw * cm->uph * sizeof(int16_t);
    residualsSizeV = cm->vpw * cm->vph * sizeof(int16_t);

    keyframe_offset = 0;
    mbOffsetY = keyframe_offset + keyframeSize;
    mbOffsetU = mbOffsetY + mbSizeY;
    mbOffsetV = mbOffsetU + mbSizeU;
    residualsY_offset = mbOffsetV + mbSizeV;
    residualsU_offset = residualsY_offset + residualsSizeY;
    residualsV_offset = residualsU_offset + residualsSizeU;

}

struct segment_yuv init_image_segment(struct c63_common* cm, int segNum)
{
	struct segment_yuv image;
	unsigned int localSegmentId = (localNodeId << 16) | (readerNodeId << 8) | (SEGMENT_ENCODER_IMAGE + segNum);

	unsigned int imageSizeY = cm->ypw * cm->yph * sizeof(uint8_t);
	unsigned int imageSizeU = cm->upw * cm->uph * sizeof(uint8_t);
	unsigned int imageSizeV = cm->vpw * cm->vph * sizeof(uint8_t);
	unsigned int imageSize = imageSizeY + imageSizeU + imageSizeV;
	unsigned int segmentSize = imageSize;

	//segmentSize = segmentSize/LOK * LOK + LOK;

	if(segmentSize < MIN_SEG_SZ) {
		segmentSize = MIN_SEG_SZ;
	}

	sci_error_t error;
	SCICreateSegment(reader_sds[segNum], &imageSegments[segNum], localSegmentId, segmentSize, SCI_NO_CALLBACK, NULL, SCI_FLAG_EMPTY, &error);
	sisci_assert(error);

	hipMalloc((void**)&cudaBuffers[segNum], 3*segmentSize);

	struct hipPointerAttribute_t attributes;
	hipPointerGetAttributes(&attributes, (void*)cudaBuffers[segNum]);

	printf("addr: %ld\n", (long unsigned int) attributes.devicePointer);

	SCIAttachPhysicalMemory(0, attributes.devicePointer, 0, segmentSize, imageSegments[segNum], SCI_FLAG_CUDA_BUFFER, &error);
	sisci_assert(error);

	unsigned int offset = 0;
	volatile void* buffer;
	buffer = SCIMapLocalSegment(imageSegments[segNum], &imageMaps[segNum], offset, imageSizeY, NULL, SCI_FLAG_READONLY_MAP, &error);
	sisci_assert(error);
	image.Y = (volatile uint8_t*) buffer;

	offset += imageSizeY;
	buffer = SCIMapLocalSegment(imageSegments[segNum], &imageMaps[segNum], offset, imageSizeU, NULL, SCI_FLAG_READONLY_MAP, &error);
	sisci_assert(error);
	image.U = (volatile uint8_t*) buffer;

	offset += imageSizeU;
	buffer = SCIMapLocalSegment(imageSegments[segNum], &imageMaps[segNum], offset, imageSizeV, NULL, SCI_FLAG_READONLY_MAP, &error);
	sisci_assert(error);
	image.V = (volatile uint8_t*) buffer;


	/*
	unsigned int offset = 0;
	image.Y = (uint8_t*) buffer + offset;
	offset += imageSizeY;
	image.U = (uint8_t*) buffer + offset;
	offset += imageSizeU;
	image.V = (uint8_t*) buffer + offset;
	offset += imageSizeV;
	*/

	SCIPrepareSegment(imageSegments[segNum], localAdapterNo, SCI_NO_FLAGS, &error);
	sisci_assert(error);

	SCISetSegmentAvailable(imageSegments[segNum], localAdapterNo, SCI_NO_FLAGS, &error);
	sisci_assert(error);

	return image;
}

void init_remote_encoded_data_segment(int segNum)
{
	unsigned int remoteSegmentId = (writerNodeId << 16) | (localNodeId << 8) | (SEGMENT_WRITER_ENCODED + segNum);

	sci_error_t error;

	// Connect to remote segment on writer
	do {
		SCIConnectSegment(writer_sds[segNum], &encodedDataSegmentsWriter[segNum], writerNodeId, remoteSegmentId, localAdapterNo,
				SCI_NO_CALLBACK, NULL, SCI_INFINITE_TIMEOUT, SCI_NO_FLAGS, &error);
	} while (error != SCI_ERR_OK);

	// Get segment size
	segmentSizeWriter = SCIGetRemoteSegmentSize(encodedDataSegmentsWriter[segNum]);

	unsigned int offset = 0;

	printf("segnum: %d\n", segNum);
	/*
	volatile void *buffer;
	buffer = SCIMapRemoteSegment(encodedDataSegmentsWriter[segNum], &encodedDataMapsRemote[segNum],
			offset, keyframeSize, NULL, SCI_NO_FLAGS, &error);
	sisci_assert(error);
	keyframe[segNum] = (volatile int*) (volatile uint8_t*) buffer;

	offset += keyframeSize;
	buffer = SCIMapRemoteSegment(encodedDataSegmentsWriter[segNum], &encodedDataMapsRemote[segNum],
				offset, mbSizeY, (void*)((uint8_t*)buffer+offset), SCI_FLAG_FIXED_MAP_ADDR, &error);
	sisci_assert(error);
	remote_mb_Y[segNum] = (struct macroblock*)(volatile uint8_t*)buffer;

	offset += mbSizeY;
	buffer = SCIMapRemoteSegment(encodedDataSegmentsWriter[segNum], &encodedDataMapsRemote[segNum],
				offset, mbSizeU, (void*)((uint8_t*)buffer+offset), SCI_FLAG_FIXED_MAP_ADDR, &error);
	sisci_assert(error);
	remote_mb_U[segNum] = (struct macroblock*)(volatile uint8_t*)buffer;

	offset += mbSizeU;
	buffer = SCIMapRemoteSegment(encodedDataSegmentsWriter[segNum], &encodedDataMapsRemote[segNum],
				offset, mbSizeV, (void*)((uint8_t*)buffer+offset), SCI_FLAG_FIXED_MAP_ADDR, &error);
	sisci_assert(error);
	remote_mb_V[segNum] = (struct macroblock*)(volatile uint8_t*)buffer;

	offset += mbSizeV;
	buffer = SCIMapRemoteSegment(encodedDataSegmentsWriter[segNum], &encodedDataMapsRemote[segNum],
				offset, residualsSizeY, (void*)((uint8_t*)buffer+offset), SCI_FLAG_FIXED_MAP_ADDR, &error);
	sisci_assert(error);
	remote_residuals_Y[segNum] = (volatile uint8_t*)buffer;

	offset += residualsSizeY;
	buffer = SCIMapRemoteSegment(encodedDataSegmentsWriter[segNum], &encodedDataMapsRemote[segNum],
				offset, residualsSizeU, (void*)((uint8_t*)buffer+offset), SCI_FLAG_FIXED_MAP_ADDR, &error);
	sisci_assert(error);
	remote_residuals_U[segNum] = (volatile uint8_t*)buffer;

	offset += residualsSizeU;
	buffer = SCIMapRemoteSegment(encodedDataSegmentsWriter[segNum], &encodedDataMapsRemote[segNum],
				offset, residualsSizeV, (void*)((uint8_t*)buffer+offset), SCI_FLAG_FIXED_MAP_ADDR, &error);
	sisci_assert(error);
	remote_residuals_V[segNum] = (volatile uint8_t*)buffer;
	*/
	/*
	volatile void *buffer;
	buffer = SCIMapRemoteSegment(encodedDataSegmentsWriter[segNum], &encodedDataMapsRemote[segNum],
			offset, keyframeSize, NULL, SCI_NO_FLAGS, &error);
	sisci_assert(error);
	keyframe[segNum] = (volatile int*) (volatile uint8_t*) buffer;

	offset += keyframeSize;
	buffer = SCIMapRemoteSegment(encodedDataSegmentsWriter[segNum], &encodedDataMapsRemote[segNum],
				offset, mbSizeY+mbSizeU+mbSizeV, NULL, SCI_NO_FLAGS, &error);
	sisci_assert(error);
	remote_mb_Y[segNum] = (struct macroblock*)(volatile uint8_t*)buffer;
	remote_mb_U[segNum] = (struct macroblock*)((volatile uint8_t*)buffer+mbSizeY);
	remote_mb_V[segNum] = (struct macroblock*)((volatile uint8_t*)buffer+mbSizeU);

	offset += mbSizeY+mbSizeU+mbSizeV;
	buffer = SCIMapRemoteSegment(encodedDataSegmentsWriter[segNum], &encodedDataMapsRemote[segNum],
				offset, residualsSizeY+residualsSizeU+residualsSizeV, NULL, SCI_NO_FLAGS, &error);
	sisci_assert(error);
	remote_residuals_Y[segNum] = (volatile uint8_t*)buffer;
	remote_residuals_U[segNum] = ((volatile uint8_t*)buffer+residualsSizeY);
	remote_residuals_V[segNum] = ((volatile uint8_t*)buffer+residualsSizeU);
	 */

	volatile void *buffer;
	buffer = SCIMapRemoteSegment(encodedDataSegmentsWriter[segNum], &encodedDataMapsRemote[segNum],
			offset, segmentSizeWriter, NULL, SCI_NO_FLAGS, &error);
	sisci_assert(error);
	keyframe[segNum] = (int*) ((uint8_t*)buffer + keyframe_offset);

	remote_mb_Y[segNum] = (struct macroblock*)((uint8_t*)buffer + mbOffsetY);
	remote_mb_U[segNum] = (struct macroblock*)((uint8_t*)buffer + mbOffsetU);
	remote_mb_V[segNum] = (struct macroblock*)((uint8_t*)buffer + mbOffsetV);

	remote_residuals_Y[segNum] = (uint8_t*)buffer + residualsY_offset;
	remote_residuals_U[segNum] = (uint8_t*)buffer + residualsU_offset;
	remote_residuals_V[segNum] = (uint8_t*)buffer + residualsV_offset;


}

void init_local_encoded_data_segment() {
	sci_error_t error;
	uint32_t localSegmentId = (localNodeId << 16) | (writerNodeId << 8) | 37;

	SCICreateSegment(writer_sds[0], &encodedDataSegmentsLocal[0], localSegmentId, segmentSizeWriter, SCI_NO_CALLBACK, NULL, SCI_NO_FLAGS, &error);
	sisci_assert(error);

	SCIPrepareSegment(encodedDataSegmentsLocal[0], localAdapterNo, SCI_FLAG_DMA_SOURCE_ONLY, &error);
	sisci_assert(error);

	volatile void *buffer = SCIMapLocalSegment(encodedDataSegmentsLocal[0], &encodedDataMapsLocal[0], 0, segmentSizeWriter, NULL, SCI_NO_FLAGS, &error);
	sisci_assert(error);

	keyframe[0] = (int*) ((uint8_t*)buffer + keyframe_offset);

	mb_Y[0] = (struct macroblock*) ((uint8_t*) buffer + mbOffsetY);
	mb_U[0] = (struct macroblock*) ((uint8_t*) buffer + mbOffsetU);
	mb_V[0] = (struct macroblock*) ((uint8_t*) buffer + mbOffsetV);
}


void init_local_encoded_data_segments() {
	sci_error_t error;
	uint32_t localSegmentId;
	void *buffer;

	unsigned int segmentSize = keyframeSize + mbSizeY + mbSizeU + mbSizeV;

	int i;
	for (i = 0; i < NUM_IMAGE_SEGMENTS; ++i) {

		localSegmentId = (localNodeId << 16) | (writerNodeId << 8) | (37 + i);

		SCICreateSegment(writer_sds[i], &encodedDataSegmentsLocal[i], localSegmentId, segmentSize, SCI_NO_CALLBACK, NULL, SCI_NO_FLAGS, &error);
		sisci_assert(error);

		printf("i: %d\n", i);

		SCIPrepareSegment(encodedDataSegmentsLocal[i], localAdapterNo, SCI_FLAG_DMA_SOURCE_ONLY, &error);
		sisci_assert(error);

		buffer = SCIMapLocalSegment(encodedDataSegmentsLocal[i], &encodedDataMapsLocal[i], 0, segmentSize, NULL, SCI_NO_FLAGS, &error);
		sisci_assert(error);

		keyframe[i] = (int*) ((uint8_t*)buffer + keyframe_offset);

		mb_Y[i] = (struct macroblock*) ((uint8_t*) buffer + mbOffsetY);
		mb_U[i] = (struct macroblock*) ((uint8_t*) buffer + mbOffsetU);
		mb_V[i] = (struct macroblock*) ((uint8_t*) buffer + mbOffsetV);
	}
}

static void cleanup_local_segment(sci_local_segment_t* segment, sci_map_t* map)
{
	sci_error_t error;

	SCISetSegmentUnavailable(*segment, localAdapterNo, SCI_NO_FLAGS, &error);
	sisci_check(error);

	SCIUnmapSegment(*map, SCI_NO_FLAGS, &error);
	sisci_check(error);

	SCIRemoveSegment(*segment, SCI_NO_FLAGS, &error);
	sisci_check(error);
}

static void cleanup_remote_segment(sci_remote_segment_t* segment, sci_map_t *map)
{
	sci_error_t error;

	SCIUnmapSegment(*map, SCI_NO_FLAGS, &error);
	sisci_check(error);

	SCIDisconnectSegment(*segment, SCI_NO_FLAGS, &error);
	sisci_check(error);
}

void cleanup_segments()
{
	int i;
	for (i = 0; i < NUM_IMAGE_SEGMENTS; ++i) {
		//cleanup_local_segment(&encodedDataSegmentsLocal[i], &encodedDataMapsLocal[i]);
		hipFree((void*)cudaBuffers[i]);
		cleanup_local_segment(&imageSegments[i], &imageMaps[i]);
		cleanup_remote_segment(&encodedDataSegmentsWriter[i], &encodedDataMapsRemote[i]);
	}

}

void receive_width_and_height(uint32_t* width, uint32_t* height)
{
	sci_error_t error;

	printf("Waiting for width and height from reader... ");
	fflush(stdout);

	uint32_t widthAndHeight[2];
	unsigned int length = 2 * sizeof(uint32_t);
	SCIWaitForDataInterrupt(interruptsFromReader[0], &widthAndHeight, &length, SCI_INFINITE_TIMEOUT,
			SCI_NO_FLAGS, &error);
	sisci_assert(error);

	*width = widthAndHeight[0];
	*height = widthAndHeight[1];
	printf("Done!\n");
}

void send_width_and_height(uint32_t width, uint32_t height) {
	sci_error_t error;

	uint32_t widthAndHeight[2] = {width, height};
	SCITriggerDataInterrupt(interruptsToWriter[0], (void*) &widthAndHeight, 2*sizeof(uint32_t), SCI_NO_FLAGS, &error);
	sisci_assert(error);
}

int wait_for_reader(int segNum)
{
	sci_error_t error;

	static unsigned int done_size = sizeof(uint8_t);
	uint8_t done;

	SCIWaitForDataInterrupt(interruptsFromReader[segNum], &done, &done_size, SCI_INFINITE_TIMEOUT, SCI_NO_FLAGS, &error);
	sisci_assert(error);

	return done;
}

void wait_for_writer(int segNum)
{
	sci_error_t error;

	do {
		SCIWaitForInterrupt(interruptsFromWriter[segNum], SCI_INFINITE_TIMEOUT, SCI_NO_FLAGS, &error);
	} while (error != SCI_ERR_OK);
}


sci_callback_action_t dma_callback(void *arg, sci_dma_queue_t dma_queue, sci_error_t status) {
	sci_callback_action_t retVal;

	if (status == SCI_ERR_OK) {
		// Send interrupt to computation node signaling that the frame has been transferred
		signal_writer(DATA_TRANSFERRED, *(int*)arg);

		retVal = SCI_CALLBACK_CONTINUE;
	}

	else {
		retVal = SCI_CALLBACK_CANCEL;
	}

	free(arg);

	return retVal;

}

void transfer_encoded_data(int keyframe_val, struct macroblock** mbs, dct_t* residuals, int segNum)
{
	//sci_error_t error;
	*keyframe[segNum] = keyframe_val;
	//memcpy(mb_Y[segNum], mbs[Y_COMPONENT], mbSizeY+mbSizeU+mbSizeV);

	/*
	dis_dma_vec_t dis_dma_vec[1];
	dis_dma_vec[0].size = sizeof(int)+mbSizeY+mbSizeU+mbSizeV;
	dis_dma_vec[0].local_offset = 0;
	dis_dma_vec[0].remote_offset = 0;
	dis_dma_vec[1].size = residualsSizeY+residualsSizeU+residualsSizeV;
	dis_dma_vec[1].local_offset = residualsY_offset;
	dis_dma_vec[1].remote_offset = residualsY_offset;
	//hipMemcpy(remote_residuals_Y[segNum], residuals->Ydct, residualsSizeY+residualsSizeU+residualsSizeV, hipMemcpyDeviceToHost);
	*/
	printf("Sending\n");
	//hipMemcpy((void*)remote_mb_Y[segNum], mbs[Y_COMPONENT], mbSizeY+mbSizeU+mbSizeV, hipMemcpyDeviceToHost);
	hipMemcpy((void*)remote_mb_Y[segNum], mbs[Y_COMPONENT], mbSizeY, hipMemcpyDeviceToHost);
	hipMemcpy((void*)remote_mb_U[segNum], mbs[U_COMPONENT], mbSizeU, hipMemcpyDeviceToHost);
	hipMemcpy((void*)remote_mb_V[segNum], mbs[V_COMPONENT], mbSizeV, hipMemcpyDeviceToHost);

	//hipMemcpy((void*)remote_residuals_Y[segNum], residuals->Ydct, residualsSizeY+residualsSizeU+residualsSizeV, hipMemcpyDeviceToHost);
	hipMemcpy((void*)remote_residuals_Y[segNum], residuals->Ydct, residualsSizeY, hipMemcpyDeviceToHost);
	hipMemcpy((void*)remote_residuals_U[segNum], residuals->Udct, residualsSizeU, hipMemcpyDeviceToHost);
	hipMemcpy((void*)remote_residuals_V[segNum], residuals->Vdct, residualsSizeV, hipMemcpyDeviceToHost);

	printf("sent\n");

	//int *arg = (int*) malloc(sizeof(int));
	//*arg = segNum;
	signal_writer(DATA_TRANSFERRED, segNum);
	//SCIStartDmaTransferVec(dmaQueues[segNum], encodedDataSegmentsLocal[segNum], encodedDataSegmentsWriter[segNum],
			//1, dis_dma_vec,	dma_callback, arg, SCI_FLAG_USE_CALLBACK, &error);
	//SCIStartDmaTransfer(dmaQueues[segNum], encodedDataSegmentsLocal[segNum], encodedDataSegmentsWriter[segNum],
		//	0, sizeof(int)+mbSizeY+mbSizeU+mbSizeV, 0, dma_callback, arg, SCI_FLAG_USE_CALLBACK, &error);
	//sisci_assert(error);
}

void wait_for_image_transfer(int segNum) {
	sci_error_t error;

	SCIWaitForDMAQueue(dmaQueues[segNum], SCI_INFINITE_TIMEOUT, SCI_NO_FLAGS, &error);
	//sisci_assert(error);
}

void signal_reader(int segNum)
{
	sci_error_t error;
	SCITriggerInterrupt(interruptsToReader[segNum], SCI_NO_FLAGS, &error);
	sisci_assert(error);
}

void signal_writer(writer_signal signal, int segNum)
{
	sci_error_t error;

	uint8_t data;

	switch (signal) {
		case ENCODING_FINISHED:
			data = 1;
			break;
		case DATA_TRANSFERRED:
			data = 0;
			break;
	}

	SCITriggerDataInterrupt(interruptsToWriter[segNum], (void*) &data, sizeof(uint8_t), SCI_NO_FLAGS, &error);
	sisci_assert(error);
}
