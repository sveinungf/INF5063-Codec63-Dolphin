#include <sisci_api.h>
#include <sisci_error.h>

#include "../common/sisci_errchk.h"
#include "sisci.h"

#define MIN_SEG_SZ 237569

// Local
static unsigned int localAdapterNo;
static unsigned int localNodeId;
sci_desc_t reader_sds[NUM_IMAGE_SEGMENTS] __attribute__((aligned(sizeof(sci_desc_t))));
sci_desc_t writer_sds[NUM_IMAGE_SEGMENTS] __attribute__((aligned(sizeof(sci_desc_t))));
uint8_t* cudaBuffers[NUM_IMAGE_SEGMENTS];

// Reader
unsigned int readerNodeId;
sci_local_data_interrupt_t interruptsFromReader[NUM_IMAGE_SEGMENTS];
sci_remote_data_interrupt_t interruptToReader;
sci_local_segment_t imageSegments[NUM_IMAGE_SEGMENTS] __attribute__((aligned(sizeof(sci_local_segment_t))));
sci_map_t imageMaps[NUM_IMAGE_SEGMENTS] __attribute__((aligned(sizeof(sci_map_t))));

// Writer
static unsigned int segmentSizeWriter;
static unsigned int writerNodeId;
static sci_local_data_interrupt_t interruptFromWriter;
static sci_remote_data_interrupt_t interruptsToWriter[NUM_IMAGE_SEGMENTS];
static sci_remote_segment_t encodedDataSegmentsWriter[NUM_IMAGE_SEGMENTS];
static sci_local_segment_t encodedDataSegmentsLocal[NUM_IMAGE_SEGMENTS];
static sci_map_t encodedDataMapsLocal[NUM_IMAGE_SEGMENTS];
static sci_dma_queue_t dmaQueues[NUM_IMAGE_SEGMENTS];

static unsigned int keyframeSize;
static unsigned int mbSizeY;
static unsigned int mbSizeU;
static unsigned int mbSizeV;
static unsigned int residualsSizeY;
static unsigned int residualsSizeU;
static unsigned int residualsSizeV;

static unsigned int keyframe_offset;
static unsigned int mbOffsetY;
static unsigned int residualsY_offset;
static unsigned int mbOffsetU;
static unsigned int residualsU_offset;
static unsigned int mbOffsetV;
static unsigned int residualsV_offset;

static int *keyframe[NUM_IMAGE_SEGMENTS];
static struct macroblock *mb_Y[NUM_IMAGE_SEGMENTS];
static struct macroblock *mb_U[NUM_IMAGE_SEGMENTS];
static struct macroblock *mb_V[NUM_IMAGE_SEGMENTS];
static dct_t *residuals_Y[NUM_IMAGE_SEGMENTS];
static dct_t *residuals_U[NUM_IMAGE_SEGMENTS];
static dct_t *residuals_V[NUM_IMAGE_SEGMENTS];


void init_SISCI(unsigned int localAdapter, unsigned int readerNode, unsigned int writerNode)
{
	localAdapterNo = localAdapter;
	readerNodeId = readerNode;
	writerNodeId = writerNode;

	sci_error_t error;

	SCIInitialize(SCI_NO_FLAGS, &error);
	sisci_assert(error);

	int i;
	for (i = 0; i < NUM_IMAGE_SEGMENTS; ++i) {
		SCIOpen(&reader_sds[i], SCI_NO_FLAGS, &error);
		sisci_assert(error);

		SCIOpen(&writer_sds[i], SCI_NO_FLAGS, &error);
		sisci_assert(error);
	}

	SCIGetLocalNodeId(localAdapterNo, &localNodeId, SCI_NO_FLAGS, &error);
	sisci_assert(error);

	unsigned int maxEntries = 1;
	for (i = 0; i < NUM_IMAGE_SEGMENTS; ++i) {
		SCICreateDMAQueue(writer_sds[i], &dmaQueues[i], localAdapterNo, maxEntries, SCI_NO_FLAGS, &error);
		sisci_assert(error);
	}

	// Interrupts from the reader
	unsigned int interruptFromReaderNo;
	for (i = 0; i < NUM_IMAGE_SEGMENTS; ++i) {
		interruptFromReaderNo = MORE_DATA_TRANSFERRED + i;
		SCICreateDataInterrupt(reader_sds[i], &interruptsFromReader[i], localAdapterNo, &interruptFromReaderNo, NULL,
				NULL, SCI_FLAG_FIXED_INTNO, &error);
		sisci_assert(error);
	}

	unsigned int interruptFromWriterNo = DATA_WRITTEN;
	SCICreateDataInterrupt(writer_sds[0], &interruptFromWriter, localAdapterNo, &interruptFromWriterNo, NULL,
					NULL, SCI_FLAG_FIXED_INTNO, &error);
	sisci_assert(error);

	// Interrupts to the reader
	printf("Connecting to interrupt on reader... ");
	fflush(stdout);
	do
	{
		SCIConnectDataInterrupt(reader_sds[0], &interruptToReader, readerNodeId, localAdapterNo,
				READY_FOR_ORIG_TRANSFER, SCI_INFINITE_TIMEOUT, SCI_NO_FLAGS, &error);
	}
	while (error != SCI_ERR_OK);
	printf("Done!\n");

	// Interrupts to the writer
	printf("Connecting to interrupt on writer... ");
	fflush(stdout);
	for (i = 0; i < NUM_IMAGE_SEGMENTS; ++i) {
		do
		{
			SCIConnectDataInterrupt(writer_sds[i], &interruptsToWriter[i], writerNodeId, localAdapterNo,
					ENCODED_FRAME_TRANSFERRED + i, SCI_INFINITE_TIMEOUT, SCI_NO_FLAGS, &error);
		}
		while (error != SCI_ERR_OK);
	}
	printf("Done!\n");
}

void cleanup_SISCI()
{
	sci_error_t error;

	SCIDisconnectDataInterrupt(interruptToReader, SCI_NO_FLAGS, &error);
	sisci_check(error);

	do {
		SCIRemoveDataInterrupt(interruptFromWriter, SCI_NO_FLAGS, &error);
	} while (error != SCI_ERR_OK);



	int i;
	for (i = 0; i < NUM_IMAGE_SEGMENTS; ++i) {
		SCIDisconnectDataInterrupt(interruptsToWriter[i], SCI_NO_FLAGS, &error);
		sisci_check(error);

		do {
			SCIRemoveDataInterrupt(interruptsFromReader[i], SCI_NO_FLAGS, &error);
		} while (error != SCI_ERR_OK);

		SCIRemoveDMAQueue(dmaQueues[i], SCI_NO_FLAGS, &error);

		SCIClose(reader_sds[i], SCI_NO_FLAGS, &error);
		sisci_check(error);

		SCIClose(writer_sds[i], SCI_NO_FLAGS, &error);
		sisci_check(error);
	}
	SCITerminate();
}


void set_sizes_offsets(struct c63_common *cm) {
    keyframeSize = sizeof(int);
    mbSizeY = cm->mb_rowsY * cm->mb_colsY * sizeof(struct macroblock);
    mbSizeU = cm->mb_rowsUV * cm->mb_colsUV * sizeof(struct macroblock);
    mbSizeV = cm->mb_rowsUV * cm->mb_colsUV * sizeof(struct macroblock);
    residualsSizeY = cm->ypw * cm->yph * sizeof(int16_t);
    residualsSizeU = cm->upw * cm->uph * sizeof(int16_t);
    residualsSizeV = cm->vpw * cm->vph * sizeof(int16_t);

    keyframe_offset = 0;
    mbOffsetY = keyframe_offset + keyframeSize;
    mbOffsetU = mbOffsetY + mbSizeY;
    mbOffsetV = mbOffsetU + mbSizeU;
    residualsY_offset = mbOffsetV + mbSizeV;
    residualsU_offset = residualsY_offset + residualsSizeY;
    residualsV_offset = residualsU_offset + residualsSizeU;

}

struct segment_yuv init_image_segment(struct c63_common* cm, int segNum)
{
	struct segment_yuv image;
	unsigned int localSegmentId = (localNodeId << 16) | (readerNodeId << 8) | (SEGMENT_ENCODER_IMAGE + segNum);

	unsigned int imageSizeY = cm->ypw * cm->yph * sizeof(uint8_t);
	unsigned int imageSizeU = cm->upw * cm->uph * sizeof(uint8_t);
	unsigned int imageSizeV = cm->vpw * cm->vph * sizeof(uint8_t);
	unsigned int imageSize = imageSizeY + imageSizeU + imageSizeV;
	unsigned int segmentSize = imageSize;

	if(segmentSize < MIN_SEG_SZ) {
		segmentSize = MIN_SEG_SZ;
	}

	sci_error_t error;
	SCICreateSegment(reader_sds[segNum], &imageSegments[segNum], localSegmentId, segmentSize, SCI_NO_CALLBACK, NULL, SCI_FLAG_EMPTY, &error);
	sisci_assert(error);

	hipMalloc((void**)&cudaBuffers[segNum], 3*segmentSize);

	struct hipPointerAttribute_t attributes;
	hipPointerGetAttributes(&attributes, (void*)cudaBuffers[segNum]);

	printf("addr: %ld\n", (long unsigned int) attributes.devicePointer);

	SCIAttachPhysicalMemory(0, attributes.devicePointer, 0, segmentSize, imageSegments[segNum], SCI_FLAG_CUDA_BUFFER, &error);
	sisci_assert(error);

	void* buffer = SCIMapLocalSegment(imageSegments[segNum], &imageMaps[segNum], 0, segmentSize, NULL, SCI_NO_FLAGS, &error);
	sisci_assert(error);

	unsigned int offset = 0;
	image.Y = (uint8_t*) buffer + offset;
	offset += imageSizeY;
	image.U = (uint8_t*) buffer + offset;
	offset += imageSizeU;
	image.V = (uint8_t*) buffer + offset;
	offset += imageSizeV;

	SCIPrepareSegment(imageSegments[segNum], localAdapterNo, SCI_NO_FLAGS, &error);
	sisci_assert(error);

	SCISetSegmentAvailable(imageSegments[segNum], localAdapterNo, SCI_NO_FLAGS, &error);
	sisci_assert(error);

	return image;
}

void init_remote_encoded_data_segment(int segNum)
{
	unsigned int remoteSegmentId = (writerNodeId << 16) | (localNodeId << 8) | (SEGMENT_WRITER_ENCODED + segNum);

	sci_error_t error;

	// Connect to remote segment on writer
	do {
		SCIConnectSegment(writer_sds[segNum], &encodedDataSegmentsWriter[segNum], writerNodeId, remoteSegmentId, localAdapterNo,
				SCI_NO_CALLBACK, NULL, SCI_INFINITE_TIMEOUT, SCI_NO_FLAGS, &error);
	} while (error != SCI_ERR_OK);

	// Get segment size
	segmentSizeWriter = SCIGetRemoteSegmentSize(encodedDataSegmentsWriter[segNum]);

}

void init_local_encoded_data_segment() {
	sci_error_t error;
	uint32_t localSegmentId = (localNodeId << 16) | (writerNodeId << 8) | 37;

	SCICreateSegment(writer_sds[0], &encodedDataSegmentsLocal[0], localSegmentId, segmentSizeWriter, SCI_NO_CALLBACK, NULL, SCI_NO_FLAGS, &error);
	sisci_assert(error);

	SCIPrepareSegment(encodedDataSegmentsLocal[0], localAdapterNo, SCI_FLAG_DMA_SOURCE_ONLY, &error);
	sisci_assert(error);

	void *buffer = SCIMapLocalSegment(encodedDataSegmentsLocal[0], &encodedDataMapsLocal[0], 0, segmentSizeWriter, NULL, SCI_NO_FLAGS, &error);
	sisci_assert(error);

	keyframe[0] = (int*) ((uint8_t*)buffer + keyframe_offset);

	mb_Y[0] = (struct macroblock*) ((uint8_t*) buffer + mbOffsetY);
	mb_U[0] = (struct macroblock*) ((uint8_t*) buffer + mbOffsetU);
	mb_V[0] = (struct macroblock*) ((uint8_t*) buffer + mbOffsetV);

	residuals_Y[0] = (dct_t*) ((uint8_t*) buffer + residualsY_offset);
	residuals_U[0] = (dct_t*) ((uint8_t*) buffer + residualsU_offset);
	residuals_V[0] = (dct_t*) ((uint8_t*) buffer + residualsV_offset);
}


void init_local_encoded_data_segments() {
	sci_error_t error;
	unsigned int localSegmentId;
	void *buffer;

	int i;
	for (i = 0; i < NUM_IMAGE_SEGMENTS; ++i) {

		localSegmentId = (localNodeId << 16) | (writerNodeId << 8) | (37 + i);

		printf("size: %d\n", segmentSizeWriter);
		SCICreateSegment(writer_sds[i], &encodedDataSegmentsLocal[i], localSegmentId, segmentSizeWriter, SCI_NO_CALLBACK, NULL, SCI_NO_FLAGS, &error);
		sisci_assert(error);

		printf("i: %d\n", i);

		SCIPrepareSegment(encodedDataSegmentsLocal[i], localAdapterNo, SCI_FLAG_DMA_SOURCE_ONLY, &error);
		sisci_assert(error);

		buffer = SCIMapLocalSegment(encodedDataSegmentsLocal[i], &encodedDataMapsLocal[i], 0, segmentSizeWriter, NULL, SCI_NO_FLAGS, &error);
		sisci_assert(error);

		keyframe[i] = (int*) ((uint8_t*)buffer + keyframe_offset);

		mb_Y[i] = (struct macroblock*) ((uint8_t*) buffer + mbOffsetY);
		mb_U[i] = (struct macroblock*) ((uint8_t*) buffer + mbOffsetU);
		mb_V[i] = (struct macroblock*) ((uint8_t*) buffer + mbOffsetV);

		residuals_Y[i] = (dct_t*) ((uint8_t*) buffer + residualsY_offset);
		residuals_U[i] = (dct_t*) ((uint8_t*) buffer + residualsU_offset);
		residuals_V[i] = (dct_t*) ((uint8_t*) buffer + residualsV_offset);
	}
}

static void cleanup_local_segment(sci_local_segment_t* segment, sci_map_t* map)
{
	sci_error_t error;

	SCISetSegmentUnavailable(*segment, localAdapterNo, SCI_NO_FLAGS, &error);
	sisci_check(error);

	SCIUnmapSegment(*map, SCI_NO_FLAGS, &error);
	sisci_check(error);

	SCIRemoveSegment(*segment, SCI_NO_FLAGS, &error);
	sisci_check(error);
}

static void cleanup_remote_segment(sci_remote_segment_t* segment)
{
	sci_error_t error;

	SCIDisconnectSegment(*segment, SCI_NO_FLAGS, &error);
	sisci_check(error);
}

void cleanup_segments()
{
	int i;
	for (i = 0; i < NUM_IMAGE_SEGMENTS; ++i) {
		cleanup_local_segment(&encodedDataSegmentsLocal[i], &encodedDataMapsLocal[i]);
		hipFree((void*)cudaBuffers[i]);
		cleanup_local_segment(&imageSegments[i], &imageMaps[i]);
		cleanup_remote_segment(&encodedDataSegmentsWriter[i]);
	}

}

void receive_width_and_height(uint32_t* width, uint32_t* height)
{
	sci_error_t error;

	printf("Waiting for width and height from reader... ");
	fflush(stdout);

	uint32_t widthAndHeight[2];
	unsigned int length = 2 * sizeof(uint32_t);
	SCIWaitForDataInterrupt(interruptsFromReader[0], &widthAndHeight, &length, SCI_INFINITE_TIMEOUT,
			SCI_NO_FLAGS, &error);
	sisci_assert(error);

	*width = widthAndHeight[0];
	*height = widthAndHeight[1];
	printf("Done!\n");
}

void send_width_and_height(uint32_t width, uint32_t height) {
	sci_error_t error;

	uint32_t widthAndHeight[2] = {width, height};
	SCITriggerDataInterrupt(interruptsToWriter[0], (void*) &widthAndHeight, 2*sizeof(uint32_t), SCI_NO_FLAGS, &error);
	sisci_assert(error);
}

int wait_for_reader(int segNum)
{
	sci_error_t error;

	static unsigned int done_size = sizeof(uint8_t);
	uint8_t done;

	SCIWaitForDataInterrupt(interruptsFromReader[segNum], &done, &done_size, SCI_INFINITE_TIMEOUT, SCI_NO_FLAGS, &error);
	sisci_assert(error);

	return done;
}

void wait_for_writer(int segNum)
{
	sci_error_t error;

	int ack;
	unsigned int length = sizeof(int);
	do {
		SCIWaitForDataInterrupt(interruptFromWriter, &ack, &length, SCI_INFINITE_TIMEOUT,
				SCI_NO_FLAGS, &error);
		sisci_assert(error);
	} while (ack != segNum);
}


sci_callback_action_t dma_callback(void *arg, sci_dma_queue_t dma_queue, sci_error_t status) {
	sci_callback_action_t retVal;

	if (status == SCI_ERR_OK) {
		// Send interrupt to computation node signaling that the frame has been transferred
		signal_writer(DATA_TRANSFERRED, *(int*)arg);

		retVal = SCI_CALLBACK_CONTINUE;
	}

	else {
		retVal = SCI_CALLBACK_CANCEL;
	}

	free(arg);

	return retVal;

}

void transfer_encoded_data(int keyframe_val, struct macroblock** mbs, dct_t* residuals, int segNum)
{
	sci_error_t error;
	*keyframe[segNum] = keyframe_val;
	memcpy(mb_Y[segNum], mbs[Y_COMPONENT], mbSizeY+mbSizeU+mbSizeV);

	memcpy(residuals_Y[segNum], residuals->base, residualsSizeY + residualsSizeU + residualsSizeV);

	int *arg = (int*) malloc(sizeof(int));
	*arg = segNum;

	SCIStartDmaTransfer(dmaQueues[segNum], encodedDataSegmentsLocal[segNum], encodedDataSegmentsWriter[segNum], 0, segmentSizeWriter, 0, dma_callback, arg, SCI_FLAG_USE_CALLBACK, &error);
	sisci_assert(error);
}

void wait_for_image_transfer(int segNum) {
	sci_error_t error;

	SCIWaitForDMAQueue(dmaQueues[segNum], SCI_INFINITE_TIMEOUT, SCI_NO_FLAGS, &error);
	sisci_assert(error);
}

void signal_reader(int segNum)
{
	sci_error_t error;

	int ack = segNum;
	SCITriggerDataInterrupt(interruptToReader, (void*) &ack, sizeof(int), SCI_NO_FLAGS, &error);

	sisci_assert(error);
}

void signal_writer(writer_signal signal, int segNum)
{
	sci_error_t error;

	uint8_t data;

	switch (signal) {
		case ENCODING_FINISHED:
			data = 1;
			break;
		case DATA_TRANSFERRED:
			data = 0;
			break;
	}

	SCITriggerDataInterrupt(interruptsToWriter[segNum], (void*) &data, sizeof(uint8_t), SCI_NO_FLAGS, &error);
	sisci_assert(error);
}
